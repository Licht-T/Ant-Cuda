#include "hip/hip_runtime.h"
#include <iostream>
#include <sstream>
#include <fstream>
#include <cmath>

#include <sys/stat.h>
#include <sys/types.h>

// includes CUDA Runtime
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <thrust/transform_reduce.h>
#include <thrust/functional.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>

#include "Constants.h"
#include "DataStructures.h"
#include "Variables.h"
#include "kernel4.h"
#include "Display.h"
#include "IO4.h"

int main(int argc, char *argv[]){

    getHoming homeOp;
    thrust::plus<int> binary_op;

    IOInit();

    initialize();

    double normalEndTime[MACRO_NUM_FOODS*2]; // 有限量の餌を採り尽くす時間
    double normalFindTime[MACRO_NUM_FOODS*2]; // 有限量の餌を採り尽くす時間
    for (int id=0; id<MACRO_NUM_FOODS; id++){
        normalEndTime[id] = 0;
        normalFindTime[id] = 0;
        normalEndTime[id+2] = 0;
        normalFindTime[id+2] = 0;
    }
    double normalSum = 0.0;

    /* for(unsigned long long int dummy=1; dummy<=MACRO_MAX_STEP; dummy++){ */
    /*     reset(pow(10,-3),50,dummy); */
    /*     display(argc,argv); */
    /*  */
    /*     for(int t=0; t<MACRO_MAX_TIME; t++){ */
    /*         calculation(); */
    /*     } */
    /* } */

    // 馬鹿ありのいない場合
    for(unsigned long long int dummy=1; dummy<=MACRO_MAX_STEP; dummy++){
        int FindTime[MACRO_NUM_FOODS]; // 有限量の餌を見つける時間
        int FindFlag[MACRO_NUM_FOODS];
        int EndTime[MACRO_NUM_FOODS]; // 有限量の餌を採り尽くす時間
        int EndFlag[MACRO_NUM_FOODS];
        for (int id=0; id<MACRO_NUM_FOODS; id++){
            EndTime[id] = 0;
            EndFlag[id] = 0;
            FindTime[id] = 0;
            FindFlag[id] = 0;
        }
        reset(1,0,dummy);
        foods[0].vol = MACRO_FOODSOURCE;
        foods[1].vol = 0;
        hipMemcpyFromSymbol(foods_d, HIP_SYMBOL(foods), sizeof(Food)*MACRO_NUM_FOODS);

        // display(argc,argv);

        for(int t=0; t<MACRO_MAX_TIME; t++){
            calculation();
            hipMemcpyFromSymbol(&foods, HIP_SYMBOL(foods_d), sizeof(Food)*MACRO_NUM_FOODS);
            if (foods[0].vol < MACRO_FOODSOURCE && FindFlag[0] == 0){
                FindTime[0] = t;
                FindFlag[0] = 1;
            }
            if (foods[1].vol < MACRO_FOODSOURCE && FindFlag[1] == 0 && EndFlag[0] == 1){
                FindTime[1] = t;
                FindFlag[1] = 1;
            }
            if (foods[0].vol < 0.04 && EndFlag[0]==0){
                foods[0].vol = 0;
                foods[1].vol = MACRO_FOODSOURCE;
                hipMemcpyFromSymbol(foods_d, HIP_SYMBOL(foods), sizeof(Food)*MACRO_NUM_FOODS);
                EndTime[0] = t;
                EndFlag[0] = 1;
            }
            if (EndFlag[0]==1 && foods[1].vol < 0.04){
                EndTime[1] = t;
                EndFlag[1] = 1;
                break;
            }
            // if(t%500==0){
            //     IOEffPoll(0,500,dummy,t);
            // }

        }
        for (int id=0; id<MACRO_NUM_FOODS; id++){
            normalEndTime[id] += EndTime[id];
            normalFindTime[id] += FindTime[id];
            normalEndTime[id+2] += EndTime[id]*EndTime[id];
            normalFindTime[id+2] += FindTime[id]*FindTime[id];
        }

        normalSum += thrust::transform_reduce(ants_d_ptr, ants_d_ptr+MACRO_NMAX, homeOp, 0, binary_op);
    }

    for (int n=0; n<=500; n+=50){
        IOEffWrite(0,n,normalSum);
        IOFoodWrite(0,n, normalFindTime, normalEndTime);
    }

    // 馬鹿アリの感受性
    for (int pw=1; pw<=7; pw++){
        // 正常アリの数
        for (int n=0; n<=450; n+=50){
            double sensor = pow(10,-pw);
            int naho = (MACRO_NMAX - n);

            double sum = 0.0;
            double EndTimeAve[MACRO_NUM_FOODS*2];
            double FindTimeAve[MACRO_NUM_FOODS*2];
            for (int id=0; id<MACRO_NUM_FOODS; id++){
                EndTimeAve[id] = 0;
                FindTimeAve[id] = 0;
                EndTimeAve[id+2] = 0;
                FindTimeAve[id+2] = 0;
            }
            for(unsigned long long int dummy=1; dummy<=MACRO_MAX_STEP; dummy++){
                int FindTime[MACRO_NUM_FOODS]; // 有限量の餌を見つける時間
                int FindFlag[MACRO_NUM_FOODS];
                int EndTime[MACRO_NUM_FOODS]; // 有限量の餌を採り尽くす時間
                int EndFlag[MACRO_NUM_FOODS];
                for (int id=0; id<MACRO_NUM_FOODS; id++){
                    EndTime[id] = 0;
                    EndFlag[id] = 0;
                    FindTime[id] = 0;
                    FindFlag[id] = 0;
                }
                reset(sensor,naho,dummy);
                foods[0].vol = MACRO_FOODSOURCE;
                foods[1].vol = 0;
                hipMemcpyFromSymbol(foods_d, HIP_SYMBOL(foods), sizeof(Food)*MACRO_NUM_FOODS);

                for(int t=1; t<=MACRO_MAX_TIME; t++){
                    calculation();
                    hipMemcpyFromSymbol(&foods, HIP_SYMBOL(foods_d), sizeof(Food)*MACRO_NUM_FOODS);
                    if (foods[0].vol < MACRO_FOODSOURCE && FindFlag[0] == 0){
                        FindTime[0] = t;
                        FindFlag[0] = 1;
                    }
                    if (foods[1].vol < MACRO_FOODSOURCE && FindFlag[1] == 0 && EndFlag[0] == 1){
                        FindTime[1] = t;
                        FindFlag[1] = 1;
                    }
                    if (foods[0].vol < 0.04 && EndFlag[0]==0){
                        foods[0].vol = 0;
                        foods[1].vol = MACRO_FOODSOURCE;
                        hipMemcpyFromSymbol(foods_d, HIP_SYMBOL(foods), sizeof(Food)*MACRO_NUM_FOODS);
                        EndTime[0] = t;
                        EndFlag[0] = 1;
                    }
                    if (EndFlag[0]==1 && foods[1].vol < 0.04){
                        EndTime[1] = t;
                        EndFlag[1] = 1;
                        break;
                    }
                    // if(t%500==0){
                    //     IOEffPoll(pw,n,dummy,t);
                    // }

                }

                for (int id=0; id<MACRO_NUM_FOODS; id++){
                    EndTimeAve[id] += EndTime[id];
                    FindTimeAve[id] += FindTime[id];
                    EndTimeAve[id+2] += EndTime[id] - FindTime[id];
                    FindTimeAve[id+2] += (EndTime[id] - FindTime[id])*(EndTime[id] - FindTime[id]);
                }
                sum += thrust::transform_reduce(ants_d_ptr, ants_d_ptr+MACRO_NMAX, homeOp, 0, binary_op);
            }
            // IOCellWrite(pw,n);
            IOEffWrite(pw,n,sum);
            IOFoodWrite(pw,n,FindTimeAve,EndTimeAve);
        }
        IOEffWrite(pw,500,normalSum);
        IOFoodWrite(pw,500, normalFindTime, normalEndTime);
    }

}

