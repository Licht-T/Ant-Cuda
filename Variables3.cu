#include "Variables3.h"

Food foods[MACRO_NUM_FOODS];
__device__ Food foods_d[MACRO_NUM_FOODS];
Food *foods_d_ptr_raw;
thrust::device_ptr<Food> foods_d_ptr;

Ant ants[MACRO_NMAX];
__device__ Ant ants_d[MACRO_NMAX];
Ant *ants_d_ptr_raw;
thrust::device_ptr<Ant> ants_d_ptr;

Cell cells[MACRO_MAX][MACRO_MAX];
__device__ Cell cells_d[MACRO_MAX][MACRO_MAX];
Cell *cells_d_ptr_raw;
thrust::device_ptr<Cell> cells_d_ptr;

Cell min_path1[MACRO_MAX][MACRO_MAX];
__device__ Cell min_path1_d[MACRO_MAX][MACRO_MAX];
Cell *min_path1_d_ptr_raw;
thrust::device_ptr<Cell> min_path1_d_ptr;

Cell min_path2[MACRO_MAX][MACRO_MAX];
__device__ Cell min_path2_d[MACRO_MAX][MACRO_MAX];
Cell *min_path2_d_ptr_raw;
thrust::device_ptr<Cell> min_path2_d_ptr;

__device__ unsigned int sort_key_d[MACRO_NMAX];
unsigned int *sort_key_d_ptr_raw;
thrust::device_ptr<unsigned int> sort_key_d_ptr;

__device__ unsigned long long int seeds_d[MACRO_NMAX];
unsigned long long int *seeds_d_ptr_raw;
thrust::device_ptr<unsigned long long int> seeds_d_ptr;

int homing = 0;
__device__ int homing_d = 0;
double x_Nest2Food[2] = {0.0, 0.0};
double y_Nest2Food[2] = {0.0, 0.0};
__device__ double x_Nest2Food_d[2];
__device__ double y_Nest2Food_d[2];

