#include "hip/hip_runtime.h"
#define _USE_MATH_DEFINES

#include "kernel11.h"

__constant__ double SENSOR2;
__constant__ int NAHO;

__device__ int selectedCounts[MACRO_NMAX];
__device__ double tmpPhero_d[MACRO_MAX][MACRO_MAX];
__device__ hiprandState rnd_state[MACRO_NMAX];

//Misc
__device__ bool isGotFood(Food& food);
__device__ double atomicAddDouble(double* address, double val);
__device__ enum Direction genDirRand(int id);
__device__ double genProbRand(int id);
__device__ int genAntNumRand(int id);
__device__ double degToRad(double a);
__device__ double dist(Cell a,Cell b);
__device__ double distCandP(Cell a,double x,double y);
__device__ bool isOppositeDir(enum Direction nestDir,enum Direction dir);
__device__ bool isOppositeDir(Cell& cell, enum Direction dir);
__device__ enum Direction selectNextDir(Cell& cell, enum Direction dir);
__device__ double hilFunc(double x,double alpha);

//Initializer
__host__ void getDevicePtrs();
__global__ void randInit();
__global__ void antsInit();
__global__ void cellsInit();
__global__ void setNest();
__global__ void setDistFromNest();
__global__ void setNestDirs();
__global__ void setNearestDirFromNest();
__global__ void setFoodsDir();

//Calculation functions
__global__ void selectAnts();
__global__ void naturalFoodDecrease();
__global__ void evapolation();
__global__ void chemotaxis();
__global__ void diffusion();
__global__ void pheroUpdate();
__global__ void min_path_init();


__host__ void calculation(){
    naturalFoodDecrease<<<1,MACRO_NUM_FOODS>>>();
    evapolation<<<MACRO_MAX,MACRO_MAX>>>();

    //sortKeyInit<<<1,MACRO_NMAX>>>();
    //thrust::sort_by_key(sort_key_d_ptr, sort_key_d_ptr + MACRO_NMAX, ants_d_ptr);

    selectAnts<<<1,MACRO_NMAX>>>();
    chemotaxis<<<1,MACRO_NMAX>>>();
    //hipMemcpyFromSymbol(cells,HIP_SYMBOL(cells_d),MACRO_MAX*MACRO_MAX*sizeof(Cell),0);
    //chemotaxis();
    //hipMemcpyToSymbol(HIP_SYMBOL(cells_d),cells,MACRO_MAX*MACRO_MAX*sizeof(Cell),0);
    diffusion<<<MACRO_MAX,MACRO_MAX>>>();
    pheroUpdate<<<MACRO_MAX,MACRO_MAX>>>();
}

// __host__ void min_path_calc(){
//     min_path<<<MACRO_MAX,MACRO_MAX>>>();
// }

//Initialize

__host__ void getDevicePtrs(){
    hipGetSymbolAddress((void**)&sort_key_d_ptr_raw, sort_key_d);
    sort_key_d_ptr = thrust::device_ptr<unsigned int>(sort_key_d_ptr_raw);

    hipGetSymbolAddress((void**)&seeds_d_ptr_raw, seeds_d);
    seeds_d_ptr = thrust::device_ptr<unsigned long long int>(seeds_d_ptr_raw);

    hipGetSymbolAddress((void**)&ants_d_ptr_raw, ants_d);
    ants_d_ptr = thrust::device_ptr<Ant>(ants_d_ptr_raw);

    hipGetSymbolAddress((void**)&cells_d_ptr_raw, cells_d);
    cells_d_ptr = thrust::device_ptr<Cell>(cells_d_ptr_raw);

    hipGetSymbolAddress((void**)&min_path1_d_ptr_raw, min_path1_d);
    min_path1_d_ptr = thrust::device_ptr<Cell>(min_path1_d_ptr_raw);

    hipGetSymbolAddress((void**)&min_path2_d_ptr_raw, min_path2_d);
    min_path2_d_ptr = thrust::device_ptr<Cell>(min_path2_d_ptr_raw);

    hipGetSymbolAddress((void**)&foods_d_ptr_raw, foods_d);
    foods_d_ptr = thrust::device_ptr<Food>(foods_d_ptr_raw);
}

__global__ void min_path_init(){
    const int i = threadIdx.x;
    const int j = blockIdx.x;

    // relative vector from nest
    double vx = cells_d[i][j].cart.x - cells_d[MACRO_NEST_Y][MACRO_NEST_X].cart.x;
    double vy = cells_d[i][j].cart.y - cells_d[MACRO_NEST_Y][MACRO_NEST_X].cart.y;

    // projection vectors to the vectors from nest to foods
    double inner[2] = {vx * x_Nest2Food_d[0] + vy * y_Nest2Food_d[0], vx * x_Nest2Food_d[1] + vy * y_Nest2Food_d[1]};
    double px[2] = {inner[0] * x_Nest2Food_d[0], inner[1] * x_Nest2Food_d[1]};
    double py[2] = {inner[0] * y_Nest2Food_d[0], inner[1] * y_Nest2Food_d[1]};

    // find the cells which are closest to the px & py : pi, pj
    int pi[2] = {0, 0};
    int pj[2] = {0, 0};
    for (int perpIndx=0; perpIndx<2; perpIndx++){
        if (inner[perpIndx] < 0.0){
            pi[perpIndx] = -1;
            pj[perpIndx] = -1;
        }
        else
        {
            Cell *nearCell=NULL;
            for(int j=0; j<MACRO_MAX; j++){
                for(int k=0; k<MACRO_MAX; k++){
                    if(distCandP(cells_d[j][k],px[perpIndx],py[perpIndx])<=sqrt(3.0)/3.0+MACRO_EPS){
                        nearCell = &cells_d[j][k];
                        break;
                    }
                }
            }
            if(nearCell==NULL){
            }
            else{
                Cell *c=NULL;
                double d = distCandP(*nearCell,px[perpIndx],py[perpIndx]);
                int ii = nearCell->i;
                int jj = nearCell->j;

                for(enum Direction dir = UP; dir<=UPLEFT; (dir<<=1) ){
                    c = getCell(cells_d,ii,jj,dir);
                    if( distCandP(*c,px[perpIndx],py[perpIndx])<d ){
                        nearCell = c;
                        d = distCandP(*nearCell,px[perpIndx],py[perpIndx]);
                    }
                }
                pi[perpIndx] = nearCell->i;
                pj[perpIndx] = nearCell->j;
            }
        }
    }

    // compare with the pheromone at the cells as the maximum pheromone
    min_path1_d[i][j].i = pi[0];
    min_path1_d[i][j].j = pj[0];
    min_path2_d[i][j].i = pi[1];
    min_path2_d[i][j].j = pj[1];
}

__global__ void randInit(){
    const int id = threadIdx.x + blockIdx.x * blockDim.x;
    hiprand_init(seeds_d[id],0,0,&rnd_state[id]);
}

__global__ void antsReset(){
    const int id = threadIdx.x + blockIdx.x * blockDim.x;
    ants_d[id].status = FORAGE;
    ants_d[id].i = MACRO_NEST_Y;
    ants_d[id].j = MACRO_NEST_X;
    ants_d[id].searchTime = 0;
    ants_d[id].dir = genDirRand(id);
    for (int i=0; i<MACRO_NUM_FOODS; i++){
        ants_d[id].homing[i] = 0;
    }
    if(id<NAHO){
        ants_d[id].ch = FOOL_CH;
    }
    else {
        ants_d[id].ch = NORMAL_CH;
    }
}

__global__ void cellsReset(){
    const int i = threadIdx.x;
    const int j = blockIdx.x;
    cells_d[i][j].phero = 0.0;
}

__global__ void cellsInit(){
    const int i = threadIdx.x;
    const int j = blockIdx.x;

    cells_d[i][j].foodNo = -1;
    cells_d[i][j].status = NORMAL_CELL;

    //Cell number initialize
    cells_d[i][j].i = i;
    cells_d[i][j].j = j;

    //Cartesian initialize
    cells_d[i][j].cart.x = (j-MACRO_CART_X_ZERO)*(sqrt(3.0)/2.0);
    cells_d[i][j].cart.y = (abs(j-MACRO_CART_X_ZERO)%2)/2.0+(i-MACRO_CART_Y_ZERO);
    //Edge initialize
    cells_d[i][j].edge = NONE;

    //Nest Dir initialize
    cells_d[i][j].nestDir = NONE;

    cells_d[i][j].distFromNest = 0.0;

    // Gauss path for food source 1
    min_path1_d[i][j].foodNo = -1;
    min_path1_d[i][j].status = NORMAL_CELL;

    //Cell number initialize
    min_path1_d[i][j].i = i;
    min_path1_d[i][j].j = j;

    //Cartesian initialize
    min_path1_d[i][j].cart.x = cells_d[i][j].cart.x;
    min_path1_d[i][j].cart.y = cells_d[i][j].cart.y;
    //Edge initialize
    min_path1_d[i][j].edge = NONE;

    //Nest Dir initialize
    min_path1_d[i][j].nestDir = NONE;

    min_path1_d[i][j].distFromNest = 0.0;

    // Gauss path for food source 1
    min_path2_d[i][j].foodNo = -1;
    min_path2_d[i][j].status = NORMAL_CELL;

    //Cell number initialize
    min_path2_d[i][j].i = i;
    min_path2_d[i][j].j = j;

    //Cartesian initialize
    min_path2_d[i][j].cart.x = cells_d[i][j].cart.x;
    min_path2_d[i][j].cart.y = cells_d[i][j].cart.y;
    //Edge initialize
    min_path2_d[i][j].edge = NONE;

    //Nest Dir initialize
    min_path2_d[i][j].nestDir = NONE;

    min_path2_d[i][j].distFromNest = 0.0;
}


__global__ void setEdges(){
    const int i = threadIdx.x;
    const int j = blockIdx.x;
    if(i==MACRO_MAX-1){ //For upper edge
        cells_d[i][j].edge |= UP;
        if(abs((j-MACRO_CART_X_ZERO)%2)==1){
            cells_d[i][j].edge |= (UPLEFT | UPRIGHT);
        }
    }
    else if(i==0){//For lower edge
        cells_d[i][j].edge |= LOW;
        if(abs((j-MACRO_CART_X_ZERO)%2)==0){
            cells_d[i][j].edge |= LOWLEFT | LOWRIGHT;
        }
    }

    if(j==0){//For left edge
        cells_d[i][j].edge |= LEFT;
    }
    else if(j==MACRO_MAX-1){//For right edge
        cells_d[i][j].edge |= RIGHT;
    }
    min_path1_d[i][j].edge = cells_d[i][j].edge;
    min_path2_d[i][j].edge = cells_d[i][j].edge;
}

__global__ void setNest(){
    const int i = threadIdx.x;
    const int j = blockIdx.x;

    Cell* c;
    if(i==MACRO_NEST_Y && j==MACRO_NEST_X){
        cells_d[MACRO_NEST_Y][MACRO_NEST_X].status |= NEST_CELL;

        min_path1_d[MACRO_NEST_Y][MACRO_NEST_X].edge = cells_d[MACRO_NEST_Y][MACRO_NEST_X].edge;
        min_path2_d[MACRO_NEST_Y][MACRO_NEST_X].edge = cells_d[MACRO_NEST_Y][MACRO_NEST_X].edge;

        for(enum Direction d = UP; d<=UPLEFT; (d<<=1) ){
            c = getCell(cells_d,MACRO_NEST_Y,MACRO_NEST_X,d);
            c->status |= NEST_NEIGHBOUR_CELL;

            c = getCell(min_path1_d,MACRO_NEST_Y,MACRO_NEST_X,d);
            c->status |= NEST_NEIGHBOUR_CELL;

            c = getCell(min_path2_d,MACRO_NEST_Y,MACRO_NEST_X,d);
            c->status |= NEST_NEIGHBOUR_CELL;
        }
    }
}

__global__ void setDistFromNest(){
    const int i = threadIdx.x;
    const int j = blockIdx.x;

    Cell *nest_c;
    nest_c = &cells_d[MACRO_NEST_Y][MACRO_NEST_X];
    double d = dist(cells_d[i][j],*nest_c);
    cells_d[i][j].distFromNest = d;
    min_path1_d[i][j].distFromNest = d;
    min_path2_d[i][j].distFromNest = d;
}

__device__ double dot(Cartesian a, Cartesian b) {
    return (a.x * b.x + a.y * b.y);
}

__device__ double cross(Cartesian a, Cartesian b) {
    return (a.x * b.y - a.y * b.x);
}

__global__ void setCriticalAngle() {
    const int i = threadIdx.x;
    const int j = blockIdx.x;


    cells_d[i][j].criticalAngle = NONE;
    min_path1_d[i][j].criticalAngle = NONE;
    min_path2_d[i][j].criticalAngle = NONE;

    if( (cells_d[i][j].status&NEAR_NEST)!=NORMAL_CELL ){
        return;
    }

    Cartesian c = cells_d[i][j].cart;
    c.x = -c.x/cells_d[i][j].distFromNest;
    c.y = -c.y/cells_d[i][j].distFromNest;

    for(enum Direction dir = UP; dir<=UPLEFT; (dir<<=1) ) {
        Cartesian d;

        switch (dir) {
            case UP:
                d.x = 0;
                d.y = 1;
                break;
            case UPRIGHT:
                d.x = 1;
                d.y = tan(M_PI/4.0);
                break;
            case LOWRIGHT:
                d.x = 1;
                d.y = -tan(M_PI/4.0);
                break;
            case LOW:
                d.x = 0;
                d.y = -1;
                break;
            case LOWLEFT:
                d.x = -1;
                d.y = -tan(M_PI/4.0);
                break;
            case UPLEFT:
                d.x = -1;
                d.y = tan(M_PI/4.0);
                break;
            default:
                break;
        }

        d.x = d.x/sqrt(dot(d,d));
        d.y = d.y/sqrt(dot(d,d));

        double dotVal = dot(c,d);
        if (dotVal<=0.3){
            cells_d[i][j].criticalAngle |= dir;
            min_path1_d[i][j].criticalAngle |= dir;
            min_path2_d[i][j].criticalAngle |= dir;
        }
    }
}


__global__ void setNearestDirFromNest(){
    const int i = threadIdx.x;
    const int j = blockIdx.x;
    Cell& c = cells_d[i][j];
    Cell& gp1 = min_path1_d[i][j];
    Cell& gp2 = min_path2_d[i][j];

    for (int itr=0; itr<6; itr++){
        c.nearestDirFromNestList[itr] = NONE;
        gp1.nearestDirFromNestList[itr] = NONE;
        gp2.nearestDirFromNestList[itr] = NONE;
    }

    enum Direction dir = UP;
    for(int itr=0; dir<=UPLEFT; itr++) {
        if ( c.criticalAngle&dir == NONE ){
            continue;
        }

        c.nearestDirFromNestList[itr] = selectNextDir(c, dir);
        gp1.nearestDirFromNestList[itr] = c.nearestDirFromNestList[itr];
        gp2.nearestDirFromNestList[itr] = c.nearestDirFromNestList[itr];
        dir<<=1;
    }
}

__global__ void setNestDirs(){
    const int i = threadIdx.x;
    const int j = blockIdx.x;

    Cell *c;

    double d = cells_d[i][j].distFromNest;
    double tmp;
    for(enum Direction dir = UP; dir<=UPLEFT; (dir<<=1) ){

        c = getCell(cells_d,i,j,dir);


        tmp=c->distFromNest;
        if( fabs(tmp-d)<MACRO_EPS ){
            cells_d[i][j].nestDir |= dir;
        }
        else if(tmp<d) {
            cells_d[i][j].nestDir = dir;
            d = tmp;
        }
        min_path1_d[i][j].nestDir = cells_d[i][j].nestDir;
        min_path2_d[i][j].nestDir = cells_d[i][j].nestDir;
    }
}

__global__ void foodsReset(){
    const int i = threadIdx.x + blockIdx.x * blockDim.x;
    foods_d[i].vol = MACRO_FOODSOURCE;
}

__global__ void setFoodsDir(){
    const int i = threadIdx.x + blockIdx.x * blockDim.x;
    const double dtheta = degToRad(MACRO_FOOD_ANGLE);

    Cell *nearCell=NULL;
    Cell *nearCellgp1=NULL;
    Cell *nearCellgp2=NULL;

    double x,y;
    x=MACRO_FOOD_DIST * cos(i*dtheta);
    y=MACRO_FOOD_DIST * sin(i*dtheta);
    for(int j=0; j<MACRO_MAX; j++){
        for(int k=0; k<MACRO_MAX; k++){
            if(distCandP(cells_d[j][k],x,y)<=sqrt(3.0)/3.0+MACRO_EPS){
                nearCell = &cells_d[j][k];
                nearCellgp1 = &min_path1_d[j][k];
                nearCellgp2 = &min_path2_d[j][k];
                break;
            }
        }
    }
    if(nearCell==NULL){
    }
    else{
        Cell *c=NULL;
        Cell *gp1=NULL;
        Cell *gp2=NULL;
        double d = distCandP(*nearCell,x,y);
        int j = nearCell->i;
        int k = nearCell->j;

        for(enum Direction dir = UP; dir<=UPLEFT; (dir<<=1) ){
            c = getCell(cells_d,j,k,dir);
            gp1 = getCell(min_path1_d,j,k,dir);
            gp2 = getCell(min_path2_d,j,k,dir);
            if( distCandP(*c,x,y)<d ){
                nearCell = c;
                nearCellgp1 = gp1;
                nearCellgp2 = gp2;
                d = distCandP(*nearCell,x,y);
            }
        }
        foods_d[i].i = nearCell->i;
        foods_d[i].j = nearCell->j;

        nearCell->foodNo = i;
        nearCell->status |= FOOD_CELL;
        nearCellgp1->foodNo = nearCell->foodNo;
        nearCellgp1->status = nearCell->status;
        nearCellgp2->foodNo = nearCell->foodNo;
        nearCellgp2->status = nearCell->status;


        for(enum Direction dir = UP; dir<=UPLEFT; (dir<<=1) ){
            c = getCell(cells_d,foods_d[i].i,foods_d[i].j,dir);
            gp1 = getCell(min_path1_d,foods_d[i].i,foods_d[i].j,dir);
            gp2 = getCell(min_path2_d,foods_d[i].i,foods_d[i].j,dir);
            c->foodNo = i;
            c->status |= FOOD_NEIGHBOUR_CELL;
            gp1->foodNo = c->foodNo;
            gp1->status = c->status;
            gp2->foodNo = c->foodNo;
            gp2->status = c->status;
        }
    }

}


//Calculation

__global__ void selectAnts(){
    const int id = threadIdx.x + blockIdx.x * blockDim.x;
    int rnd =  genAntNumRand(id);
    atomicAdd(&selectedCounts[rnd], 1);
}

__global__ void sortKeyInit(){
    const int id = threadIdx.x + blockIdx.x * blockDim.x;
    sort_key_d[id] = hiprand(&rnd_state[id]);
    //printf("id:%d,%u\n",id,sort_key_d[id]);
}

__global__ void diffusion(){
    const int i = blockIdx.x;
    const int j = threadIdx.x;

    double tmp = 0.0;
    for (enum Direction dir = UP; dir<=UPLEFT; (dir<<=1) ){
        tmp += getCell(cells_d,i,j,dir)->phero;
    }
    tmpPhero_d[i][j] = cells_d[i][j].phero+MACRO_DIFFE*(tmp/6.0-cells_d[i][j].phero);
}

__global__ void pheroUpdate(){
    const int i = blockIdx.x;
    const int j = threadIdx.x;

    cells_d[i][j].phero = tmpPhero_d[i][j];
}

__global__ void naturalFoodDecrease(){
    const int id = threadIdx.x + blockIdx.x * blockDim.x;
    // 常に一定量で増加する。
    foods_d[id].vol = foods_d[id].vol + MACRO_REC;
    if (foods_d[id].vol > 100.0)
        foods_d[id].vol = 100.0;
}

__global__ void evapolation(){
    const int i = blockIdx.x;
    const int j = threadIdx.x;
    cells_d[i][j].phero *= (1.0-MACRO_EVAPOLATION_CONST);
}


__global__ void chemotaxis(){
    const int id = threadIdx.x + blockIdx.x * blockDim.x;
    Ant *ant = &(ants_d[id]);

    for(int dummy=0; dummy<selectedCounts[id]; dummy++){
        ant->searchTime++;

        int i = ant->i;
        int j = ant->j;
        enum Direction dir = ant->dir;
        enum Direction nestDir = cells_d[i][j].nestDir;

        double leftPhero, frontPhero, rightPhero;

        Cell *leftCell  = getCell(cells_d,i,j,left(dir));
        Cell *frontCell = getCell(cells_d,i,j,dir);
        Cell *rightCell = getCell(cells_d,i,j,right(dir));

        if(
                ant->searchTime>=MACRO_MAX_SEARCH_TIME
                && ant->status!=EMERGENCY
          ){
            ant->status = EMERGENCY;
        }

        if(ant->status==GOHOME){
            atomicAddDouble(&(cells_d[i][j].phero),MACRO_EMI*MACRO_ENEST);
        }
        __threadfence();
        if(ant->status==RANDOM_SEARCH){
            leftPhero = 1.0;
            frontPhero = 1.0;
            rightPhero = 1.0;
        }
        else {
            leftPhero = leftCell->phero;
            frontPhero = frontCell->phero;
            rightPhero = rightCell->phero;
        }

        if( (ant->status==GOHOME || ant->status==EMERGENCY) && isOppositeDir(cells_d[i][j], dir)){

            enum Direction nextDir = cells_d[i][j].nearestDirFromNestList[dirToNum(dir)];

            if( nextDir == left(dir) ){
                ant->dir = left(dir);
                frontCell = leftCell;
            }
            else if( nextDir == right(dir) ){
                ant->dir = right(dir);
                frontCell = rightCell;
            }
            else{
                if(genProbRand(id)<=0.5){
                    ant->dir = right(dir);
                    frontCell = rightCell;
                }
                else{
                    ant->dir = left(dir);
                    frontCell = leftCell;
                }
            }
            ant->i = frontCell->i;
            ant->j = frontCell->j;
        }
        else{
            double s1,s2,s3,s12,t,tot,rand;
            if(ant->ch == NORMAL_CH){
                t = MACRO_HIL_CONST;
            }
            else{
                t = SENSOR2*MACRO_HIL_CONST;
            }

            s1=hilFunc(leftPhero,t);
            s2=hilFunc(frontPhero,t);
            s3=hilFunc(rightPhero,t);
            /*
               if(s1<MACRO_EPS && s2<MACRO_EPS && s3<MACRO_EPS){
               s1=1.0;
               s2=1.0;
               s3=1.0;
               }
               */
            tot = s1+s2+s3;
            s1/=tot;
            s2/=tot;

            s12=s1+s2;

            rand=genProbRand(id);

            if(rand<=s1){
                ant->dir = left(dir);
                ant->i   = leftCell->i;
                ant->j   = leftCell->j;
            }
            else if(rand<=s12){
                ant->i   = frontCell->i;
                ant->j   = frontCell->j;
            }
            else{
                ant->dir = right(dir);
                ant->i   = rightCell->i;
                ant->j   = rightCell->j;
            }

        }

        if( (cells_d[ant->i][ant->j].status&NEAR_FOOD)!=NORMAL_CELL
                &&  foods_d[  cells_d[ant->i][ant->j].foodNo  ].vol>=0.1
                &&  (ant->status != GOHOME && ant->status != EMERGENCY) ){
            //atomicAddDouble(&(foods_d[  cells_d[ant->i][ant->j].foodNo  ].vol),-MACRO_UNIT);
            //ant->status = GOHOME;
            //ant->searchTime = 0;
            int fNo = cells_d[ant->i][ant->j].foodNo;

            if(isGotFood(foods_d[fNo])){
                ant->status = GOHOME;
                ant->searchTime = 0;
                ant->_foodNo = fNo;
                ant->dir = left(left(left(dir)));
            }
        }
        __threadfence();

        if( (cells_d[ant->i][ant->j].status&NEAR_NEST)!=NORMAL_CELL
                &&  (ant->status == GOHOME || ant->status == EMERGENCY)){
            if(ant->status == GOHOME){
                ant->homing[ant->_foodNo]++;
                //atomicAddDouble(&(cells_d[i][j].phero),MACRO_EMI*MACRO_ENEST);
            }
            ant->status = FORAGE;
            ant->searchTime = 0;
            ant->dir = genDirRand(id);
            ant->i   = MACRO_NEST_Y;
            ant->j   = MACRO_NEST_X;
        }
    }
    selectedCounts[id] = 0;
}


//DataHandler
__device__ __host__  enum Direction operator<<(enum Direction d, int i){
    return static_cast<enum Direction>(static_cast<int>(d)<<i);
}

__device__ __host__  enum Direction operator>>(enum Direction d, int i){
    return static_cast<enum Direction>(static_cast<int>(d)>>i);
}

__device__ __host__  enum Direction operator|(enum Direction d1, enum Direction d2){
    return static_cast<enum Direction>(static_cast<int>(d1)|static_cast<int>(d2));
}
__device__ __host__  enum Direction operator&(enum Direction d1, enum Direction d2){
    return static_cast<enum Direction>(static_cast<int>(d1)&static_cast<int>(d2));
}

__device__ __host__  enum Direction& operator|=(enum Direction& d1, enum Direction d2){
    d1 = (d1 | d2);
    return d1;
}

__device__ __host__  enum Direction& operator&=(enum Direction& d1, enum Direction d2){
    d1 = (d1 & d2);
    return d1;
}

__device__ __host__  enum Direction& operator<<=(enum Direction& d1, int i){
    d1 = (d1 << i);
    return d1;
}

__device__ __host__  enum Direction& operator>>=(enum Direction& d1, int i){
    d1 = (d1 >> i);
    return d1;
}

__device__ __host__  bool operator<=(enum Direction d1, enum Direction d2){
    return (static_cast<int>(d1) <= static_cast<int>(d2));
}







__device__ __host__  enum CELLStatus operator<<(enum CELLStatus d, int i){
    return static_cast<enum CELLStatus>(static_cast<int>(d)<<i);
}

__device__ __host__  enum CELLStatus operator>>(enum CELLStatus d, int i){
    return static_cast<enum CELLStatus>(static_cast<int>(d)>>i);
}

__device__ __host__  enum CELLStatus operator|(enum CELLStatus d1, enum CELLStatus d2){
    return static_cast<enum CELLStatus>(static_cast<int>(d1)|static_cast<int>(d2));
}
__device__ __host__  enum CELLStatus operator&(enum CELLStatus d1, enum CELLStatus d2){
    return static_cast<enum CELLStatus>(static_cast<int>(d1)&static_cast<int>(d2));
}

__device__ __host__  enum CELLStatus& operator|=(enum CELLStatus& d1, enum CELLStatus d2){
    d1 = (d1 | d2);
    return d1;
}

__device__ __host__  enum CELLStatus& operator&=(enum CELLStatus& d1, enum CELLStatus d2){
    d1 = (d1 & d2);
    return d1;
}



__device__ __host__ __forceinline__ enum Direction left(enum Direction dir){
    if(dir == UP){
        return UPLEFT;
    }
    else{
        return (dir >> 1)&ALL_DIR;
    }
}

__device__ __host__ __forceinline__ enum Direction right(enum Direction dir){
    if(dir == UPLEFT){
        return UP;
    }
    else{
        return (dir << 1)&ALL_DIR;
    }
}

__device__ __host__ __forceinline__ Cell* up(Cell cells[MACRO_MAX][MACRO_MAX],int i,int j){
    if( (cells[i][j].edge&UP)!=NONE ){
        return &cells[0][j];
    }
    else{
        return &cells[i+1][j];
    }
}

__device__ __host__ __forceinline__ Cell* upright(Cell cells[MACRO_MAX][MACRO_MAX],int i,int j){
    int ii,jj;
    if( (cells[i][j].edge&UPRIGHT)!=NONE ){
        jj = 0;
        if(abs(j-MACRO_CART_X_ZERO)%2==0){
            ii = i;
        }
        else{
            ii = i+1;
            if(ii==MACRO_MAX){
                ii = 0;
            }
        }
    }
    else{
        jj = j+1;
        if(abs(j-MACRO_CART_X_ZERO)%2==0){
            ii = i;
        }
        else{
            ii = i+1;
        }
    }
    return &cells[ii][jj];
}

__device__ __host__ __forceinline__ Cell* lowright(Cell cells[MACRO_MAX][MACRO_MAX],int i,int j){

    int ii,jj;

    if( (cells[i][j].edge&LOWRIGHT)!=NONE ){
        jj = 0;
        if(abs(j-MACRO_CART_X_ZERO)%2==0){
            ii = i-1;
            if(ii<0){
                ii=MACRO_MAX-1;
            }
        }
        else{
            ii = i;
        }
    }
    else{
        jj = j+1;
        if(abs(j-MACRO_CART_X_ZERO)%2==0){
            ii = i-1;
        }
        else{
            ii = i;
        }
    }
    return &cells[ii][jj];
}

__device__ __host__ __forceinline__ Cell* low(Cell cells[MACRO_MAX][MACRO_MAX],int i,int j){
    if( (cells[i][j].edge&LOW)!=NONE ){
        return &cells[MACRO_MAX-1][j];
    }
    else{
        return &cells[i-1][j];
    }
}

__device__ __host__ __forceinline__ Cell* lowleft(Cell cells[MACRO_MAX][MACRO_MAX],int i,int j){
    int ii,jj;

    if( (cells[i][j].edge&LOWLEFT)!=NONE ){
        jj = MACRO_MAX-1;
        if(abs(j-MACRO_CART_X_ZERO)%2==0){
            ii = i-1;
            if(ii<0){
                ii = MACRO_MAX-1;
            }
        }
        else{
            ii = i;
        }
    }
    else{
        jj = j-1;
        if(abs(j-MACRO_CART_X_ZERO)%2==0){
            ii = i-1;
        }
        else{
            ii=i;
        }
    }
    return &cells[ii][jj];
}

__device__ __host__ __forceinline__ Cell* upleft(Cell cells[MACRO_MAX][MACRO_MAX],int i,int j){
    int ii,jj;
    if( (cells[i][j].edge&UPLEFT)!=NONE ){
        jj = MACRO_MAX-1;

        if(abs(j-MACRO_CART_X_ZERO)%2==0){
            ii = i;
        }
        else{
            ii= i+1;
            if(ii==MACRO_MAX){
                ii=0;
            }
        }
    }
    else{
        jj = j-1;
        if(abs(j-MACRO_CART_X_ZERO)%2==0){
            ii = i;
        }
        else{
            ii = i+1;
        }
    }
    return &cells[ii][jj];
}

__device__ __host__ Cell* getCell(Cell cells[MACRO_MAX][MACRO_MAX],int i,int j, enum Direction dir){

    switch (dir){
        case UP:
            return up(cells,i,j);
        case UPRIGHT:
            return upright(cells,i,j);
        case LOWRIGHT:
            return lowright(cells,i,j);
        case LOW:
            return low(cells,i,j);
        case LOWLEFT:
            return lowleft(cells,i,j);
        case UPLEFT:
            return upleft(cells,i,j);
        default:
            return NULL;
    }
}

__device__ __host__ int dirToNum(enum Direction dir){
    switch (dir){
        case UP:
            return 0;
        case UPRIGHT:
            return 1;
        case LOWRIGHT:
            return 2;
        case LOW:
            return 3;
        case LOWLEFT:
            return 4;
        case UPLEFT:
            return 5;
        default:
            return -1;
    }
}



//Misc
__device__ __forceinline__ bool isGotFood(Food& food){
    unsigned long long int* address_as_ull =
        (unsigned long long int*)(&(food.vol));
    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        if(__longlong_as_double(assumed)<0.1){
            return false;
        }
        old = atomicCAS(address_as_ull, assumed,__double_as_longlong(-MACRO_UNIT + __longlong_as_double(assumed)));
    } while (assumed != old);
    return true;
}

__device__ __forceinline__ double atomicAddDouble(double* address, double val){
    unsigned long long int* address_as_ull =
        (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                __double_as_longlong(val +
                    __longlong_as_double(assumed)));
    } while (assumed != old);
    return __longlong_as_double(old);
}

__device__ __forceinline__ enum Direction genDirRand(int id){
    return static_cast<enum Direction>(1 << (hiprand(&rnd_state[id])%6));
}

__device__ __forceinline__ double genProbRand(int id){
    return hiprand_uniform_double(&rnd_state[id]);
}

__device__ __forceinline__ int genAntNumRand(int id){
    return hiprand(&rnd_state[id])%MACRO_NMAX;
}

__device__ __forceinline__ double degToRad(double a) {
    return a * M_PI / 180.0;
}

__device__ __forceinline__ double dist(Cell a,Cell b){
    return sqrt( (a.cart.x - b.cart.x)*(a.cart.x - b.cart.x)
            + (a.cart.y - b.cart.y)*(a.cart.y - b.cart.y) );
}

__device__ __forceinline__ double distCandP(Cell a,double x,double y){
    return sqrt( (a.cart.x - x)*(a.cart.x - x)
            + (a.cart.y - y)*(a.cart.y - y) );
}

__device__ __forceinline__ bool isOppositeDir(enum Direction nestDir,enum Direction dir){
    //If theta = 60 deg., this is OK.
    if( (dir&nestDir)        !=NONE
            ||  (left(dir)&nestDir)  !=NONE
            ||  (right(dir)&nestDir) !=NONE){
        return false;
    }
    else{
        return true;
    }
}

__device__ __forceinline__ bool isOppositeDir(Cell& cell, enum Direction dir){
    if ( (cell.criticalAngle & dir)==dir ){
        return true;
    }
    else{
        return false;
    }
}

__device__ __forceinline__ enum Direction selectNextDir(Cell& cell, enum Direction dir){
    int rightCount = 0;
    int leftCount  = 0;
    for (enum Direction currentDir=right(dir); currentDir!=dir; currentDir=right(currentDir)){
        if( (cell.criticalAngle & currentDir)!=currentDir ){
            break;
        }
        rightCount++;
    }

    for (enum Direction currentDir=left(dir); currentDir!=dir; currentDir=left(currentDir)){
        if( (cell.criticalAngle & currentDir)!=currentDir ){
            break;
        }
        leftCount++;
    }

    if ( rightCount < leftCount ){
        return right(dir);
    }
    else if ( rightCount > leftCount ){
        return left(dir);
    }
    else{
        return NONE;
    }
}

__device__ __forceinline__ double hilFunc(double x,double alpha){
    return pow(alpha*x+0.05,10);
}

__host__ void initialize(){
    getDevicePtrs();

    //antsInit<<<MACRO_NMAX,1>>>();
    cellsInit<<<MACRO_MAX,MACRO_MAX>>>();

    setEdges<<<MACRO_MAX,MACRO_MAX>>>();
    setNest<<<MACRO_MAX,MACRO_MAX>>>();
    setDistFromNest<<<MACRO_MAX,MACRO_MAX>>>();

    setCriticalAngle<<<MACRO_MAX,MACRO_MAX>>>();
    setNearestDirFromNest<<<MACRO_MAX,MACRO_MAX>>>();

    setNestDirs<<<MACRO_MAX,MACRO_MAX>>>();
    setFoodsDir<<<MACRO_NUM_FOODS,1>>>();

    // addition by Shiraishi
    hipMemcpyFromSymbol(&cells, HIP_SYMBOL(cells_d), sizeof(Cell)*MACRO_MAX*MACRO_MAX);
    hipMemcpyFromSymbol(&foods, HIP_SYMBOL(foods_d), sizeof(Food)*MACRO_NUM_FOODS);
    hipMemcpyFromSymbol(&x_Nest2Food, HIP_SYMBOL(x_Nest2Food_d), sizeof(double)*2);
    hipMemcpyFromSymbol(&y_Nest2Food, HIP_SYMBOL(y_Nest2Food_d), sizeof(double)*2);
    // printf("Nest, %lf, %lf\n", cells[MACRO_NEST_Y][MACRO_NEST_X].cart.x, cells[MACRO_NEST_Y][MACRO_NEST_X].cart.y);
    for (int i=0; i<MACRO_NUM_FOODS; i++){
        // printf("Foods %d, %lf, %lf\n", i, cells[foods[i].i][foods[i].j].cart.x, cells[foods[i].i][foods[i].j].cart.y);
        x_Nest2Food[i] = cells[foods[i].i][foods[i].j].cart.x - cells[MACRO_NEST_Y][MACRO_NEST_X].cart.x;
        y_Nest2Food[i] = cells[foods[i].i][foods[i].j].cart.y - cells[MACRO_NEST_Y][MACRO_NEST_X].cart.y;
        double norm_inv = 1.0 / sqrt( x_Nest2Food[i]*x_Nest2Food[i] + y_Nest2Food[i]*y_Nest2Food[i] );
        x_Nest2Food[i] = x_Nest2Food[i] * norm_inv;
        y_Nest2Food[i] = y_Nest2Food[i] * norm_inv;
    }
    hipMemcpyToSymbol(HIP_SYMBOL(x_Nest2Food_d), &x_Nest2Food, sizeof(double)*2);
    hipMemcpyToSymbol(HIP_SYMBOL(y_Nest2Food_d), &y_Nest2Food, sizeof(double)*2);
    min_path_init<<<MACRO_MAX,MACRO_MAX>>>();
}

__host__ void reset(double sensor,int naho,unsigned long long int step){
    hipMemcpyToSymbol(HIP_SYMBOL(SENSOR2),&sensor,sizeof(double),0);
    hipMemcpyToSymbol(HIP_SYMBOL(NAHO),&naho,sizeof(int),0);

    //initialize();
    //antsInit<<<MACRO_NMAX,1>>>();
    //cellsInit<<<MACRO_MAX,MACRO_MAX>>>();

    //setEdges<<<MACRO_MAX,MACRO_MAX>>>();
    //setNest<<<MACRO_MAX,MACRO_MAX>>>();
    //setDistFromNest<<<MACRO_MAX,MACRO_MAX>>>();

    //setNestDirs<<<MACRO_MAX,MACRO_MAX>>>();
    //setFoodsDir<<<MACRO_NUM_FOODS,1>>>();

    srand(MACRO_RND_SEED+step);

    thrust::host_vector<unsigned long long int> seeds_vec_h(MACRO_NMAX);
    std::generate(seeds_vec_h.begin(), seeds_vec_h.end(), rand);
    thrust::copy(seeds_vec_h.begin(), seeds_vec_h.end(), seeds_d_ptr);
    randInit<<<MACRO_NMAX,1>>>();

    antsReset<<<MACRO_NMAX,1>>>();
    cellsReset<<<MACRO_MAX,MACRO_MAX>>>();
    foodsReset<<<MACRO_NUM_FOODS,1>>>();
}
