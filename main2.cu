#include "hip/hip_runtime.h"
#include <iostream>
#include <sstream>
#include <fstream>
#include <cmath>
#include <vector>
#include <algorithm>

#include <sys/stat.h>
#include <sys/types.h>

// includes CUDA Runtime
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <thrust/transform_reduce.h>
#include <thrust/functional.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>

#include "Constants.h"
#include "DataStructures.h"
#include "Variables.h"
#include "kernel2.h"
//#include "kernel.h"
#include "Display.h"
#include "IO2.h"

int main(int argc, char *argv[]){


    getHoming homeOp;
    thrust::plus<int> binary_op;

    IOInit();
    std::string step = toString(MACRO_MAX_STEP);
    std::string angle = toString(MACRO_FOOD_ANGLE);

    initialize();

    double normalEndTime[MACRO_NUM_FOODS]; // 有限量の餌を採り尽くす時間
    double normalFindTime[MACRO_NUM_FOODS]; // 有限量の餌を採り尽くす時間
    for (int id=0; id<MACRO_NUM_FOODS; id++){
        normalEndTime[id] = 0;
        normalFindTime[id] = 0;
    }
    double normalSum = 0.0;
    // std::vector< std::vector<double> > FoodsAmount(MACRO_MAX_TIME, std::vector<double>(MACRO_NUM_FOODS, 0));

    /* for(unsigned long long int dummy=1; dummy<=MACRO_MAX_STEP; dummy++){ */
    /*     reset(pow(10,-3),50,dummy); */
    /*     display(argc,argv); */
    /*  */
    /*     for(int t=0; t<MACRO_MAX_TIME; t++){ */
    /*         calculation(); */
    /*     } */
    /* } */

    // 馬鹿ありのいない場合
    for(unsigned long long int dummy=1; dummy<=MACRO_MAX_STEP; dummy++){
        // std::string pwstr = toString(0);
        // std::string nstr = toString(0);
        // std::string estr = toString(dummy);
        // std::string fool_num_foodamount = std::string(step+"steps_"+angle+"deg_"+pwstr+"_"+nstr+"_"+estr+"_"+"_foodamount.dat");
        // std::ofstream ofsfoodamount(fool_num_foodamount.c_str());
        // FoodsAmount = std::vector< std::vector<double> >(MACRO_MAX_TIME, std::vector<double>(MACRO_NUM_FOODS, 0));
        int FindTime[MACRO_NUM_FOODS]; // 有限量の餌を見つける時間
        int FindFlag[MACRO_NUM_FOODS];
        int EndTime[MACRO_NUM_FOODS]; // 有限量の餌を採り尽くす時間
        int EndFlag[MACRO_NUM_FOODS];
        for (int id=0; id<MACRO_NUM_FOODS; id++){
            EndTime[id] = 0;
            EndFlag[id] = 0;
            FindTime[id] = 0;
            FindFlag[id] = 0;
        }
        int FinFoodNum = 0;
        reset(1,0,dummy);

        // display(argc,argv);

        for(int t=0; t<MACRO_MAX_TIME; t++){
            calculation();
            // if(t%500==0){
            //     IOEffPoll(0,500,dummy,t);
            // }

            hipMemcpyFromSymbol(&foods, HIP_SYMBOL(foods_d), sizeof(Food)*MACRO_NUM_FOODS);
            for (int id=0; id<MACRO_NUM_FOODS; id++){
                // FoodsAmount[t][id] = foods[id].vol;
                if (foods[id].vol < MACRO_FOODSOURCE && FindFlag[id]==0)
                {
                    FindTime[id] = t;
                    FindFlag[id] = 1;
                }
                if (foods[id].vol < 0.04 && EndFlag[id] == 0){
                    EndTime[id] = t;
                    EndFlag[id] = 1;
                    FinFoodNum++;
                }
            }
            if (FinFoodNum == MACRO_NUM_FOODS)
                break;
        }
        for (int id=0; id<MACRO_NUM_FOODS; id++){
            normalEndTime[id] += EndTime[id];
            normalFindTime[id] += FindTime[id];
        }

        // IOFoodAmountWrite(ofsfoodamount, FoodsAmount);
       normalSum += thrust::transform_reduce(ants_d_ptr, ants_d_ptr+MACRO_NMAX, homeOp, 0, binary_op);
    }

    for (int n=0; n<=500; n+=50){
       IOEffWrite(0,n,normalSum);
       IOFoodWrite(0,n, normalFindTime, normalEndTime);
    }

    // 馬鹿アリの感受性
    for (int pw=1; pw<=7; pw++){
        // 正常アリの数
        for (int n=0; n<=450; n+=50){
            double sensor = pow(10,-pw);
            int naho = (MACRO_NMAX - n);

            double sum = 0.0;
            double EndTimeAve[MACRO_NUM_FOODS];
            double FindTimeAve[MACRO_NUM_FOODS];
            for (int id=0; id<MACRO_NUM_FOODS; id++){
                EndTimeAve[id] = 0;
                FindTimeAve[id] = 0;
            }
            for(unsigned long long int dummy=1; dummy<=MACRO_MAX_STEP; dummy++){
                // std::string pwstr = toString(pw);
                // std::string nstr = toString(n);
                // std::string estr = toString(dummy);
                // std::string fool_num_foodamount = std::string(step+"steps_"+angle+"deg_"+pwstr+"_"+nstr+"_"+estr+"_"+"_foodamount.dat");
                // std::ofstream ofsfoodamount(fool_num_foodamount.c_str());
                // FoodsAmount = std::vector< std::vector<double> >(MACRO_MAX_TIME, std::vector<double>(MACRO_NUM_FOODS, 0));
                int FindTime[MACRO_NUM_FOODS]; // 有限量の餌を見つける時間
                int FindFlag[MACRO_NUM_FOODS];
                int EndTime[MACRO_NUM_FOODS]; // 有限量の餌を採り尽くす時間
                int EndFlag[MACRO_NUM_FOODS];
                for (int id=0; id<MACRO_NUM_FOODS; id++){
                    EndTime[id] = 0;
                    EndFlag[id] = 0;
                    FindTime[id] = 0;
                    FindFlag[id] = 0;
                }
                int FinFoodNum = 0;
                reset(sensor,naho,dummy);

                for(int t=1; t<=MACRO_MAX_TIME; t++){
                    calculation();
                    // if(t%500==0){
                    //     IOEffPoll(pw,n,dummy,t);
                    // }

                    hipMemcpyFromSymbol(&foods, HIP_SYMBOL(foods_d), sizeof(Food)*MACRO_NUM_FOODS);
                    for (int id=0; id<MACRO_NUM_FOODS; id++){
                        // FoodsAmount[t][id] = foods[id].vol;
                        if (foods[id].vol < MACRO_FOODSOURCE && FindFlag[id]==0)
                        {
                            FindTime[id] = t;
                            FindFlag[id] = 1;
                        }
                        if (foods[id].vol < 0.04 && EndFlag[id] == 0){
                            EndTime[id] = t;
                            EndFlag[id] = 1;
                            FinFoodNum++;
                        }
                    }
                    if (FinFoodNum == MACRO_NUM_FOODS)
                        break;
                }

                for (int id=0; id<MACRO_NUM_FOODS; id++){
                    EndTimeAve[id] += EndTime[id];
                    FindTimeAve[id] += FindTime[id];
                }
                // IOFoodAmountWrite(ofsfoodamount, FoodsAmount);
               sum += thrust::transform_reduce(ants_d_ptr, ants_d_ptr+MACRO_NMAX, homeOp, 0, binary_op);
            }
            // IOCellWrite(pw,n);
            IOEffWrite(pw,n,sum);
            IOFoodWrite(pw,n,FindTimeAve,EndTimeAve);
        }
        IOEffWrite(pw,500,normalSum);
        IOFoodWrite(pw,500, normalFindTime, normalEndTime);
    }

}

