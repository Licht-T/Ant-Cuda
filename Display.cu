#include "hip/hip_runtime.h"
#include "Display.h"

void displayFunc();
void init();
void reshape(int w,int h);
void idle();

float getMaxPhero(){
    float max = -1;
    for (int i=0; i<MAX; i++){
        for (int j=0; j<MAX; j++){
            float p = cells[i][j].phero;
            if (max<p){
                max=p;
            }
        }
    }
    return max;
}


void getHeatMapColor(float value, float cl[3]){
    const int NUM_COLORS = 4;
    static float color[NUM_COLORS][3] = { {0,0,1}, {0,1,0}, {1,1,0}, {1,0,0} };
    // A static array of 4 colors:  (blue,   green,  yellow,  red) using {r,g,b} for each.

    int idx1;        // |-- Our desired color will be between these two indexes in "color".
    int idx2;        // |
    float fractBetween = 0;  // Fraction between "idx1" and "idx2" where our value is.

    if(value <= 0){
        idx1 = idx2 = 0;
    }    // accounts for an input <=0
    else if(value >= 1){
        idx1 = idx2 = NUM_COLORS-1;
    }    // accounts for an input >=0
    else{
        value = value * (NUM_COLORS-1);        // Will multiply value by 3.
        idx1  = floor(value);                  // Our desired color will be after this index.
        idx2  = idx1+1;                        // ... and before this index (inclusive).
        fractBetween = value - float(idx1);    // Distance between the two indexes (0-1).
    }

    cl[0] = (color[idx2][0] - color[idx1][0])*fractBetween + color[idx1][0];
    cl[1] = (color[idx2][1] - color[idx1][1])*fractBetween + color[idx1][1];
    cl[2] = (color[idx2][2] - color[idx1][2])*fractBetween + color[idx1][2];
}

void display(int argc, char *argv[]){
    glutInitWindowSize(600, 600);
    glutInit(&argc, argv);
    glutInitDisplayMode(GLUT_SINGLE | GLUT_RGBA | GLUT_DOUBLE);
    glutCreateWindow("Ants");
    glutDisplayFunc(displayFunc);
    glutIdleFunc(idle);
    glutReshapeFunc(reshape);
    init();
    glutMainLoop();
}

void idle(void)
{
    glutPostRedisplay();
}

void glDrawHexLine(double x,double y, double r){
    glBegin(GL_LINE_LOOP);
    glColor3f(1.0, 1.0, 1.0);
    for (int i=0; i<6; i++){
        glVertex2f(x + r*cos(i*M_PI/3), y + r*sin(i*M_PI/3));
    }
    glEnd();
}

void glDrawHex(double x,double y, double r, float rgb[3]){
    glBegin(GL_TRIANGLE_FAN);
    glColor3f(rgb[0], rgb[1], rgb[2]);
    glVertex2f(x, y);
    for (int i=0; i<6; i++){
        glVertex2f(x + r*cos(i*M_PI/3), y + r*sin(i*M_PI/3));
    }
    glEnd();
}

void glDrawNest(double x,double y,double r){
    glBegin(GL_TRIANGLE_FAN);
    glColor3f(0.0, 1.0, 0.0);
    glVertex2f(x, y);
    for (int i=0; i<=6; i++){
        glVertex2f(x + r*cos(i*M_PI/3), y + r*sin(i*M_PI/3));
    }
    glEnd();
}

void glDrawFood(double x,double y,double r){
    glBegin(GL_TRIANGLE_FAN);
    glColor3f(1.0, 0.0, 0.0);
    glVertex2f(x, y);
    for (int i=0; i<=6; i++){
        glVertex2f(x + r*cos(i*M_PI/3), y + r*sin(i*M_PI/3));
    }
    glEnd();
}

void drawCells(){
    float maxPhero = getMaxPhero();
    for(int i=1; i<MAX; i++){
        for(int j=1; j<MAX; j++){
            double x = cells[i][j].cart.x;
            double y = cells[i][j].cart.y;
            float p = cells[i][j].phero;
            float rgb[3];
            if ( (cells[i][j].status&NEAR_NEST) != NORMAL_CELL ){
                glDrawNest(x,y,1/sqrt(3));
            }
            else if ( (cells[i][j].status&NEAR_FOOD) != NORMAL_CELL ){
                glDrawFood(x,y,1/sqrt(3));
            }
            else{
                getHeatMapColor(p/maxPhero, rgb);
                glDrawHex(x,y,1/sqrt(3),rgb);
            }
        }
    }
}

void drawAnts(){
    for(int k=1; k<NMAX; k++){
        int i,j;
        enum AntStatus s = ants[k].status;
        i = ants[k].i;
        j = ants[k].j;
        double x,y;
        x = cells[i][j].cart.x;
        y = cells[i][j].cart.y;

        glPointSize(5);
        glBegin(GL_POINTS);
        switch (s){
            case FORAGE:
                glColor3f(0.0, 1.0, 0.0);
                break;
            case GOHOME:
                glColor3f(1.0, 0.0, 0.0);
                break;
            case EMERGENCY:
                glColor3f(1.0, 1.0, 1.0);
                break;
            default:
                exit(1);
        }
        glVertex2f(x , y);
        glEnd();
    }
}

void displayFunc(void){
    glClear(GL_COLOR_BUFFER_BIT);
    calculation();
    hipMemcpyFromSymbol(cells,HIP_SYMBOL(cells_d),MAX*MAX*sizeof(Cell),0);
    hipMemcpyFromSymbol(ants,HIP_SYMBOL(ants_d),NMAX*sizeof(Ant),0);
    drawCells();
    drawAnts();

    glutSwapBuffers();
}

void init(){
    glClearColor(0.0, 0.0, 0.0, 1.0);
}

void reshape(int w, int h){
    glMatrixMode(GL_PROJECTION);
    glViewport(0, 0, w, h);
    glLoadIdentity();
    gluOrtho2D(-(CART_X_ZERO+1),(MAX-CART_X_ZERO)+1,-(CART_Y_ZERO+1),(MAX-CART_Y_ZERO)+1);
}
