#include "hip/hip_runtime.h"
#include <iostream>
#include <sstream>
#include <fstream>
#include <cmath>

#include <sys/stat.h>
#include <sys/types.h>

// includes CUDA Runtime
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <thrust/transform_reduce.h>
#include <thrust/inner_product.h>
#include <thrust/functional.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>

#include "Constants.h"
#include "DataStructures2.h"
#include "Variables2.h"
#include "kernel10.h"
#include "Display10.h"
#include "IO10.h"

int main(int argc, char *argv[]){

    getHoming homeOp;
    thrust::plus<int> binary_op;
    thrust::plus<double> binary_op_add;

    IOInit();

    initialize();

    double normalSum = 0.0;

    int access[MACRO_NUM_FOODS] = {0, 0};
    double prob[5] = {0.0, 0.0, 0.0, 0.0, 0.0}; // 0:どちらにもアクセスしていない, 1:片方にアクセス, 2:両方にアクセス, 3:id=0にアクセスしている状態, 4:id=1にアクセスしている状態
    double probNormal[5] = {0.0, 0.0, 0.0, 0.0, 0.0};
    double foodspre[MACRO_NUM_FOODS] = {0.0, 0.0};

    /* for(unsigned long long int dummy=1; dummy<=MACRO_MAX_STEP; dummy++){ */
    /*     reset(pow(10,-3),50,dummy); */
    /*     display(argc,argv); */
    /*  */
    /*     for(int t=0; t<MACRO_MAX_TIME; t++){ */
    /*         calculation(); */
    /*     } */
    /* } */

    double gp1norm = sqrt(thrust::inner_product(gauss_path1_d_ptr, gauss_path1_d_ptr + MACRO_MAX*MACRO_MAX, gauss_path1_d_ptr, 0.0, binary_op_add, cell_phero_mul()));
    double gp2norm = sqrt(thrust::inner_product(gauss_path2_d_ptr, gauss_path2_d_ptr + MACRO_MAX*MACRO_MAX, gauss_path2_d_ptr, 0.0, binary_op_add, cell_phero_mul()));

    // 馬鹿ありのいない場合
    for(unsigned long long int dummy=1; dummy<=MACRO_MAX_STEP; dummy++){
        reset(1,0,dummy);

        // display(argc,argv);

        int ti = 0;
        double conv[10][3]= {{0.0, 0.0, 0.0}, {0.0, 0.0, 0.0}, {0.0, 0.0, 0.0}, {0.0, 0.0, 0.0}, {0.0, 0.0, 0.0}, {0.0, 0.0, 0.0}, {0.0, 0.0, 0.0}, {0.0, 0.0, 0.0}, {0.0, 0.0, 0.0}, {0.0, 0.0, 0.0}};
        for (int id=0; id<MACRO_NUM_FOODS; id++)
            foodspre[id] = MACRO_FOODSOURCE;
        for(int t=0; t<MACRO_MAX_TIME; t++){
            calculation();
            hipMemcpyFromSymbol(&foods, HIP_SYMBOL(foods_d), sizeof(Food)*MACRO_NUM_FOODS);
            for (int id=0; id<MACRO_NUM_FOODS; id++){
                if( foods[id].vol < foodspre[id] + MACRO_REC ){
                    access[id] = 1;
                    probNormal[3+id] += 1;
                }
                else{
                    access[id] = 0;
                }
                foodspre[id] = foods[id].vol;
            }
            probNormal[access[0]+access[1]] += 1;
            // if(t%500==0){
            //     IOEffPoll(0,500,dummy,t);
            // }
            if(t%1000==0){
                conv[ti][0] = thrust::inner_product(cells_d_ptr, cells_d_ptr + MACRO_MAX*MACRO_MAX, gauss_path1_d_ptr, 0.0, binary_op_add, cell_phero_mul()) / gp1norm;
                conv[ti][1] = thrust::inner_product(cells_d_ptr, cells_d_ptr + MACRO_MAX*MACRO_MAX, gauss_path2_d_ptr, 0.0, binary_op_add, cell_phero_mul()) / gp2norm;
                conv[ti][2] = sqrt( thrust::inner_product(cells_d_ptr, cells_d_ptr + MACRO_MAX*MACRO_MAX, cells_d_ptr, 0.0, binary_op_add, cell_phero_mul()) );
                ti++;

                // IOCellWrite(0, 500, dummy, t);
            }

        }

        IOPheroStateWrite(0, 500, dummy, conv);
        normalSum += thrust::transform_reduce(ants_d_ptr, ants_d_ptr+MACRO_NMAX, homeOp, 0, binary_op);
    }

    for (int n=0; n<=500; n+=50){
        IOEffWrite(0,n,normalSum);
        // IOProbWrite(0,n,probNormal);
    }

    // 馬鹿アリの感受性
    for (int pw=1; pw<=7; pw++){
        // 正常アリの数
        for (int n=0; n<=450; n+=50){
            double sensor = pow(10,-pw);
            int naho = (MACRO_NMAX - n);

            double sum = 0.0;
            for (int p=0; p<5; p++)
                prob[p] = 0;

            for(unsigned long long int dummy=1; dummy<=MACRO_MAX_STEP; dummy++){
                reset(sensor,naho,dummy);

                // std::string pwstr = toString(pw);
                // std::string nstr = toString(n);
                // std::string anglestr = toString(MACRO_FOOD_ANGLE);
                // std::string samplestr = toString(dummy);

                // std::string pherostatedata(path+"cell_"+anglestr+"deg_10e-"+pwstr+"_"+nstr+"normal"+"_sampleNo"+samplestr+"_phero_state.dat");
                // std::ofstream pherostate_fs(pherostatedata.c_str());

                for (int id=0; id<MACRO_NUM_FOODS; id++)
                    foodspre[id] = MACRO_FOODSOURCE;

                int ti = 0;
                double conv[10][3]= {{0.0, 0.0, 0.0}, {0.0, 0.0, 0.0}, {0.0, 0.0, 0.0}, {0.0, 0.0, 0.0}, {0.0, 0.0, 0.0}, {0.0, 0.0, 0.0}, {0.0, 0.0, 0.0}, {0.0, 0.0, 0.0}, {0.0, 0.0, 0.0}, {0.0, 0.0, 0.0}};
                for(int t=1; t<=MACRO_MAX_TIME; t++){
                    calculation();
                    hipMemcpyFromSymbol(&foods, HIP_SYMBOL(foods_d), sizeof(Food)*MACRO_NUM_FOODS);

                    for (int id=0; id<MACRO_NUM_FOODS; id++){
                        if( foods[id].vol < foodspre[id] + MACRO_REC ){
                            access[id] = 1;
                            prob[3+id] += 1;
                        }
                        else{
                            access[id] = 0;
                        }
                        foodspre[id] = foods[id].vol;
                    }
                    prob[access[0]+access[1]] += 1;
                    // if(t%500==0){
                    //     IOEffPoll(pw,n,dummy,t);
                    // }
                    if(t%1000==0){
                        conv[ti][0] = thrust::inner_product(cells_d_ptr, cells_d_ptr + MACRO_MAX*MACRO_MAX, gauss_path1_d_ptr, 0.0, binary_op_add, cell_phero_mul()) / gp1norm;
                        conv[ti][1] = thrust::inner_product(cells_d_ptr, cells_d_ptr + MACRO_MAX*MACRO_MAX, gauss_path2_d_ptr, 0.0, binary_op_add, cell_phero_mul()) / gp2norm;
                        conv[ti][2] = sqrt( thrust::inner_product(cells_d_ptr, cells_d_ptr + MACRO_MAX*MACRO_MAX, cells_d_ptr, 0.0, binary_op_add, cell_phero_mul()) );
                        ti++;

                        // IOCellWrite(pw, n, dummy, t);
                    }

                }

                IOPheroStateWrite(pw, n, dummy, conv);
                sum += thrust::transform_reduce(ants_d_ptr, ants_d_ptr+MACRO_NMAX, homeOp, 0, binary_op);
            }
            // IOCellWrite(pw,n);
           IOEffWrite(pw,n,sum);
           IOProbWrite(pw,n,prob);
        }
        IOEffWrite(pw,500,normalSum);
        IOProbWrite(pw,500,probNormal);
    }

}

