#include "hip/hip_runtime.h"
#include <iostream>
#include <sstream>
#include <fstream>
#include <cmath>

#include <sys/stat.h>
#include <sys/types.h>

// includes CUDA Runtime
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <thrust/transform_reduce.h>
#include <thrust/inner_product.h>
#include <thrust/functional.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>

#include "Constants.h"
#include "DataStructures2.h"
#include "Variables3.h"
#include "kernel11.h"
#include "Display11.h"
#include "IO11.h"

int main(int argc, char *argv[]){

    getHoming homeOp;
    thrust::plus<int> binary_op;
    thrust::plus<double> binary_op_add;

    IOInit();

    initialize();

    double normalSum = 0.0;

    int access[MACRO_NUM_FOODS] = {0, 0};
    double prob[5] = {0.0, 0.0, 0.0, 0.0, 0.0}; // 0:どちらにもアクセスしていない, 1:片方にアクセス, 2:両方にアクセス, 3:id=0にアクセスしている状態, 4:id=1にアクセスしている状態
    double probNormal[5] = {0.0, 0.0, 0.0, 0.0, 0.0};
    double foodspre[MACRO_NUM_FOODS] = {0.0, 0.0};

    /* for(unsigned long long int dummy=1; dummy<=MACRO_MAX_STEP; dummy++){ */
    /*     reset(pow(10,-3),50,dummy); */
    /*     display(argc,argv); */
    /*  */
    /*     for(int t=0; t<MACRO_MAX_TIME; t++){ */
    /*         calculation(); */
    /*     } */
    /* } */

    double gp1norm = sqrt(thrust::inner_product(min_path1_d_ptr, min_path1_d_ptr + MACRO_MAX*MACRO_MAX, min_path1_d_ptr, 0.0, binary_op_add, cell_phero_mul()));
    double gp2norm = sqrt(thrust::inner_product(min_path2_d_ptr, min_path2_d_ptr + MACRO_MAX*MACRO_MAX, min_path2_d_ptr, 0.0, binary_op_add, cell_phero_mul()));

    hipMemcpyFromSymbol(&cells, HIP_SYMBOL(cells_d), sizeof(Cell)*MACRO_MAX*MACRO_MAX);
    // 個々のセルがどの最短経路上のセルに対応しているかの情報を持つ
    hipMemcpyFromSymbol(&min_path1, HIP_SYMBOL(min_path1_d), sizeof(Cell)*MACRO_MAX*MACRO_MAX);
    hipMemcpyFromSymbol(&min_path2, HIP_SYMBOL(min_path2_d), sizeof(Cell)*MACRO_MAX*MACRO_MAX);

    size_t mp_len[MACRO_NUM_FOODS] = {0, 0};
    std::vector< std::vector< std::vector<int> > > gp;
    gp.resize(MACRO_NUM_FOODS);

    Cell (*p_mp[2])[MACRO_MAX];
    p_mp[0] = min_path1;
    p_mp[1] = min_path2;

    for (int fi=0; fi<MACRO_NUM_FOODS; fi++){
        for (int ci=0; ci<MACRO_MAX; ci++){
            for (int cj=0; cj<MACRO_MAX; cj++){
                Cell * c1 = &p_mp[fi][ci][cj];
                // Cell * c1 = &p_mp[ci][cj];
                std::cout << fi << " : " << ci*MACRO_MAX+(cj+1) << " / " << MACRO_MAX*MACRO_MAX << std::endl;
                if (c1->i == ci & c1->j == cj){
                    std::cout << "on" << std::endl;
                    std::vector<int> v(ci, cj);
                    gp[fi].push_back(v);
                }
            }
        }
        mp_len[fi] = gp[fi].size();
    }

    std::vector< std::vector< std::vector<double> > > pathlen(MACRO_NUM_FOODS, std::vector< std::vector<double> >(int(MACRO_MAX_TIME/1000), std::vector<double>(MACRO_MAX_STEP, 0.0)));

    // // 馬鹿ありのいない場合
    // for(unsigned long long int dummy=1; dummy<=MACRO_MAX_STEP; dummy++){
    //     reset(1,0,dummy);

    //     // display(argc,argv);

    //     int ti = 0;
    //     for (int id=0; id<MACRO_NUM_FOODS; id++)
    //         foodspre[id] = MACRO_FOODSOURCE;
    //     for(int t=0; t<MACRO_MAX_TIME; t++){
    //         calculation();
    //         hipMemcpyFromSymbol(&foods, HIP_SYMBOL(foods_d), sizeof(Food)*MACRO_NUM_FOODS);
    //         for (int id=0; id<MACRO_NUM_FOODS; id++){
    //             if( foods[id].vol < foodspre[id] + MACRO_REC ){
    //                 access[id] = 1;
    //                 probNormal[3+id] += 1;
    //             }
    //             else{
    //                 access[id] = 0;
    //             }
    //             foodspre[id] = foods[id].vol;
    //         }
    //         probNormal[access[0]+access[1]] += 1;
    //         // if(t%500==0){
    //         //     IOEffPoll(0,500,dummy,t);
    //         // }
    //         if(t%1000==0){
    //             hipMemcpyFromSymbol(&cells, HIP_SYMBOL(cells_d), sizeof(Cell)*MACRO_MAX*MACRO_MAX);
    //             std::vector< std::vector< std::vector<double> > > mpxyp;//(2, std::vector< std::vector< int> > (mp_len[0], std::vector<int> (3, 0)) );
    //             mpxyp.resize(MACRO_NUM_FOODS);
    //             for(int fi=0; fi<MACRO_NUM_FOODS; fi++){
    //                 mpxyp[fi].resize(mp_len[fi]);
    //                 for(int l=0; l<mp_len[fi]; l++){
    //                     mpxyp[fi][l].resize(4, 0.0);
    //                 }
    //             }
    //             // パスを得るアルゴリズム!!
    //             for (int fi=0; fi<MACRO_NUM_FOODS; fi++){
    //                 // パスを得る
    //                 for (int l_i=0; l_i<mp_len[fi]; l_i++){
    //                     for (int ci=0; ci<MACRO_MAX; ci++){
    //                         for (int cj=0; cj<MACRO_MAX; cj++){
    //                             Cell * c = &p_mp[fi][ci][cj];
    //                             if (c->i == gp[fi][l_i][0] & c->j == gp[fi][l_i][1])
    //                                 if ( cells[c->i][c->j].phero > mpxyp[fi][l_i][2] ){
    //                                     mpxyp[fi][l_i][0] = c->cart.x;
    //                                     mpxyp[fi][l_i][1] = c->cart.x;
    //                                     mpxyp[fi][l_i][2] = cells[c->i][c->j].phero;
    //                                 }
    //                         }
    //                     }
    //                 }
    //                 // 長さを測る
    //                 for (int l_i=1; l_i<mp_len[fi]; l_i++){
    //                     mpxyp[fi][l_i][3] += sqrt( (mpxyp[fi][l_i-1][0] - mpxyp[fi][l_i][0])*(mpxyp[fi][l_i-1][0] - mpxyp[fi][l_i][0]) + (mpxyp[fi][l_i-1][1] - mpxyp[fi][l_i][1])*(mpxyp[fi][l_i-1][1] - mpxyp[fi][l_i][1]) );
    //                 }
    //                 pathlen[fi][t%1000][dummy-1] = mpxyp[fi][mp_len[fi]-1][3];
    //             }

    //             IOCellWrite(0, 500, dummy, t);
    //             IOPheroPathWrite(0, 500, dummy, t, mpxyp);
    //         }

    //     }

    //     normalSum += thrust::transform_reduce(ants_d_ptr, ants_d_ptr+MACRO_NMAX, homeOp, 0, binary_op);
    // }
    // IOPheroPathLengthWrite(0, 500, pathlen);
    // for (int n=0; n<=500; n+=50){
    //     IOEffWrite(0,n,normalSum);
    //     // IOProbWrite(0,n,probNormal);
    // }

    // 馬鹿アリの感受性
    // for (int pw=1; pw<=7; pw++)
    {
        int pw = 2;
        // 正常アリの数
        // for (int n=0; n<=450; n+=50)
        {
            int n = 200;
            double sensor = pow(10,-pw);
            int naho = (MACRO_NMAX - n);

            double sum = 0.0;
            for (int p=0; p<5; p++)
                prob[p] = 0;

            std::vector< std::vector< std::vector<double> > > pathlen(MACRO_NUM_FOODS, std::vector< std::vector<double> >(int(MACRO_MAX_TIME/1000), std::vector<double>(MACRO_MAX_STEP, 0.0)));

            for(unsigned long long int dummy=1; dummy<=10; dummy++){
            // for(unsigned long long int dummy=1; dummy<=MACRO_MAX_STEP; dummy++){
                reset(sensor,naho,dummy);

                // std::string pwstr = toString(pw);
                // std::string nstr = toString(n);
                // std::string anglestr = toString(MACRO_FOOD_ANGLE);
                // std::string samplestr = toString(dummy);

                // std::string pherostatedata(path+"cell_"+anglestr+"deg_10e-"+pwstr+"_"+nstr+"normal"+"_sampleNo"+samplestr+"_phero_state.dat");
                // std::ofstream pherostate_fs(pherostatedata.c_str());

                for (int id=0; id<MACRO_NUM_FOODS; id++)
                    foodspre[id] = MACRO_FOODSOURCE;

                int ti = 0;

                for(int t=1; t<=MACRO_MAX_TIME; t++){
                    calculation();
                    hipMemcpyFromSymbol(&foods, HIP_SYMBOL(foods_d), sizeof(Food)*MACRO_NUM_FOODS);

                    for (int id=0; id<MACRO_NUM_FOODS; id++){
                        if( foods[id].vol < foodspre[id] + MACRO_REC ){
                            access[id] = 1;
                            prob[3+id] += 1;
                        }
                        else{
                            access[id] = 0;
                        }
                        foodspre[id] = foods[id].vol;
                    }
                    prob[access[0]+access[1]] += 1;
                    // if(t%500==0){
                    //     IOEffPoll(pw,n,dummy,t);
                    // }
                    if(t%1000==0){
                        hipMemcpyFromSymbol(&cells, HIP_SYMBOL(cells_d), sizeof(Cell)*MACRO_MAX*MACRO_MAX);
                        std::vector< std::vector< std::vector<double> > > mpxyp;//(2, std::vector< std::vector< int> > (mp_len[0], std::vector<int> (3, 0)) );
                        mpxyp.resize(MACRO_NUM_FOODS);
                        for(int fi=0; fi<MACRO_NUM_FOODS; fi++){
                            mpxyp[fi].resize(mp_len[fi]);
                            for(int l=0; l<mp_len[fi]; l++){
                                mpxyp[fi][l].resize(4, 0.0);
                            }
                        }
                        // パスを得るアルゴリズム!!
                        for (int fi=0; fi<MACRO_NUM_FOODS; fi++){
                            // パスを得る
                            for (int l_i=0; l_i<mp_len[fi]; l_i++){
                                for (int ci=0; ci<MACRO_MAX; ci++){
                                    for (int cj=0; cj<MACRO_MAX; cj++){
                                        Cell * c = &p_mp[fi][ci][cj];
                                        if (c->i == gp[fi][l_i][0] & c->j == gp[fi][l_i][1])
                                            if ( cells[c->i][c->j].phero > mpxyp[fi][l_i][2] ){
                                                mpxyp[fi][l_i][0] = c->cart.x;
                                                mpxyp[fi][l_i][1] = c->cart.x;
                                                mpxyp[fi][l_i][2] = cells[c->i][c->j].phero*10;
                                            }
                                    }
                                }
                            }
                            // 長さを測る
                            for (int l_i=1; l_i<mp_len[fi]; l_i++){
                                mpxyp[fi][l_i][3] += sqrt( (mpxyp[fi][l_i-1][0] - mpxyp[fi][l_i][0])*(mpxyp[fi][l_i-1][0] - mpxyp[fi][l_i][0]) + (mpxyp[fi][l_i-1][1] - mpxyp[fi][l_i][1])*(mpxyp[fi][l_i-1][1] - mpxyp[fi][l_i][1]) );
                            }
                            pathlen[fi][t%1000][dummy-1] = mpxyp[fi][mp_len[fi]-1][3];
                        }

                        IOCellWrite(pw, n, dummy, t);
                        IOPheroPathWrite(pw, n, dummy, t, mpxyp);
                    }

                }

                IOPheroPathLengthWrite(pw, n, pathlen);
                sum += thrust::transform_reduce(ants_d_ptr, ants_d_ptr+MACRO_NMAX, homeOp, 0, binary_op);
            }
            // IOCellWrite(pw,n);
           IOEffWrite(pw,n,sum);
           IOProbWrite(pw,n,prob);
        }
        IOEffWrite(pw,500,normalSum);
        IOProbWrite(pw,500,probNormal);
    }
}
