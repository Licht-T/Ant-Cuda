#include "hip/hip_runtime.h"
#include "IO.h"

int pw_old;

const std::string constHeader("Constants.h");
std::string path;
std::string fool_num_plot;
std::ofstream *ofs;

template <int max> struct getHomingFood{
    int operator()(int i){
        static const int n = max-1;
        static thrust::plus<int> binary_op;
        if(n==i){
            return thrust::transform_reduce(ants_d_ptr, ants_d_ptr+NMAX, getHomingWithFoodNum<n>(), 0, binary_op);;
        }
        else{
            return getHomingFood<n>()(i);
        }
    }
};

template <> struct getHomingFood<0>{
    int operator()(int i){
        return -1;
    }
};

template <AntCharacter ch,int max> struct getHomingTypeAndFood{
    int operator()(int i){
        static const int n = max-1;
        static thrust::plus<int> binary_op;
        if(n==i){
            return thrust::transform_reduce(ants_d_ptr, ants_d_ptr+NMAX, getHomingWithTypeAndFoodNum<ch,n>(), 0, binary_op);;
        }
        else{
            return getHomingTypeAndFood<ch,n>()(i);
        }
    }
};

template <AntCharacter ch> struct getHomingTypeAndFood<ch,0>{
    int operator()(int i){
        return -1;
    }
};

struct getPheroAroundFood{
    __device__ double operator()(const Food food) const{
        int i = food.i;
        int j = food.j;
        double sum = 0.0;
        sum += cells_d[i][j].phero;

        Cell *c=NULL;
        for(enum Direction dir = UP; dir<=UPLEFT; (dir<<=1) ){
            c = getCell(cells_d,i,j,dir);
            sum += c->phero;
        }
        return sum/7.0;
    }
};

template <class T> std::string toString(T t){
    static std::ostringstream ss;
    ss << (T)t;

    std::string str(ss.str());
    ss.str("");
    ss.clear(std::stringstream::goodbit);

    return str;
}

void IOInit(){	

    struct stat st;

    pw_old = -1;

    std::string nmax = toString(NMAX);
    std::string max = toString(MAX);

    std::string initDir(nmax+"ants_"+max+"x"+max+"cells");
    if(stat(initDir.c_str(), &st) != 0){
        mkdir(initDir.c_str(), 0775);
    }

    std::string fnum = toString(NUM_FOODS);

    std::string fNumDir(initDir+"/"+fnum+"foodnum");
    if(stat(fNumDir.c_str(), &st) != 0){
        mkdir(fNumDir.c_str(), 0775);
    }

    std::string fsource = toString(FOODSOURCE);
    std::string fdist = toString(FOOD_DIST);

    std::string fCondDir(fNumDir+"/"+fsource+"initfvol_"+fdist+"fdist");
    if(stat(fCondDir.c_str(), &st) != 0){
        mkdir(fCondDir.c_str(), 0775);
    }

    std::string step = toString(MAX_STEP);
    std::string angle = toString(FOOD_ANGLE);

    std::string stepAngleDir(fCondDir+"/"+step+"steps_"+angle+"deg");
    if(stat(stepAngleDir.c_str(), &st) != 0){
        mkdir(stepAngleDir.c_str(), 0775);
    }

    path = std::string(stepAngleDir+"/");
    fool_num_plot = std::string(path+step+"steps_"+angle+"deg.dat");

    ofs = new std::ofstream(fool_num_plot.c_str());

    std::ifstream constifs(constHeader.c_str());
    std::ofstream constofs((path+constHeader).c_str());
    constofs << constifs.rdbuf() << std::flush;
}

void IOEffWrite(int pw, int n, double sum){
    if(pw_old<pw){
        pw_old = pw;
        (*ofs) << std::endl;
    }
    (*ofs)  << pw << " "
        << n  << " "
        << (sum/(MAX_STEP))/(MAX_TIME-1000)
        << std::endl;
}

void IOCellWrite(int pw, int n){
    std::string pwstr = toString(pw);
    std::string nstr = toString(n);
    std::string anglestr = toString(FOOD_ANGLE);

    std::string celldata(path+"cell_"+anglestr+"deg_10e-"+pwstr+"_"+nstr+"normal"+".dat");
    std::ofstream cellfs(celldata.c_str());

    hipMemcpyFromSymbol(cells,HIP_SYMBOL(cells_d),MAX*MAX*sizeof(Cell),0);
    for(int i=0; i<MAX; i++){
        for(int j=0; j<MAX; j++){
            cellfs << cells[j][i].cart.x << " "
                << cells[j][i].cart.y << " "
                << cells[j][i].phero
                << std::endl;
        }
        cellfs << std::endl;
    }
}

void IOEffPoll(int pw, int n, int sample, int t){

    static getHomingWithType<NORMAL_CH> normalOp;
    static getHomingWithType<FOOL_CH> ahoOp;
    static thrust::plus<int> binary_op;
    static int homingFoods[NUM_FOODS];
    static int foolHomingFoods[NUM_FOODS];
    static int normalHomingFoods[NUM_FOODS];
    static getHomingFood<NUM_FOODS> homingFoodFunctor;
    static getHomingTypeAndFood<FOOL_CH,NUM_FOODS> foolHomingFoodFunctor;
    static getHomingTypeAndFood<NORMAL_CH,NUM_FOODS> normalHomingFoodFunctor;
    static thrust::host_vector<double> phero_h(NUM_FOODS);
    static thrust::device_vector<double> phero_d(NUM_FOODS);	

    std::string stepstr = toString(MAX_STEP);
    std::string anglestr = toString(FOOD_ANGLE);

    std::string pwstr = toString(pw);
    std::string nstr = toString(n);
    std::string samplestr = toString(sample);

    std::string celldata(path+"food_"+anglestr+"deg_10e-"+pwstr+"_"+nstr+"normal"+"_sampleNo"+samplestr+"_of_"+stepstr+".dat");
    std::ofstream pollfs(celldata.c_str(),std::ios::out | std::ios::app);

    int nor = thrust::transform_reduce(ants_d_ptr, ants_d_ptr+NMAX, normalOp, 0, binary_op);
    int aho = thrust::transform_reduce(ants_d_ptr, ants_d_ptr+NMAX, ahoOp, 0, binary_op);

    thrust::transform(foods_d_ptr, foods_d_ptr+NUM_FOODS, phero_d.begin(), getPheroAroundFood());
    thrust::copy(phero_d.begin(), phero_d.end(), phero_h.begin());

    pollfs  << t << " ";

    for (int i=0; i<NUM_FOODS; i++){
        //homingFoods[i] = homingFoodFunctor(i);
        foolHomingFoods[i] = foolHomingFoodFunctor(i);
        normalHomingFoods[i] = normalHomingFoodFunctor(i);
        homingFoods[i]=foolHomingFoods[i]+normalHomingFoods[i];
    }

    for (int i=0; i<NUM_FOODS; i++){
        pollfs  << homingFoods[i]
            << " ";
    }
    pollfs  << nor      << " "
        << aho      << " "
        << nor/(double)n << " "
        << aho/(double)(NMAX-n) << " "
        << (nor+aho)<< " ";
    for (int i=0; i<NUM_FOODS; i++){
        pollfs << normalHomingFoods[i] << " ";
    }
    for (int i=0; i<NUM_FOODS; i++){
        pollfs << foolHomingFoods[i] << " ";
    }
    for (int i=0; i<NUM_FOODS; i++){
        pollfs << normalHomingFoods[i]/(double)n << " ";
    }
    for (int i=0; i<NUM_FOODS; i++){
        pollfs << foolHomingFoods[i]/(double)(NMAX-n) << " ";
    }
    pollfs << (nor+aho) << " ";
    for (int i=0; i<NUM_FOODS; i++){
        pollfs << phero_h[i] << " ";
    }
    pollfs << std::endl;
}
