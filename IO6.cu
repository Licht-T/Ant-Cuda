#include "hip/hip_runtime.h"
#include "IO6.h"

int pw_old;

const std::string constHeader("Constants.h");
std::string path;
std::string fool_num_plot;
std::ofstream *ofs;
std::string fool_num_plot_prob;
std::ofstream *ofsProb;
std::string fool_num_food;
std::ofstream *ofsfood;

template <int max> struct getHomingFood{
    int operator()(int i){
        static const int n = max-1;
        static thrust::plus<int> binary_op;
        if(n==i){
            return thrust::transform_reduce(ants_d_ptr, ants_d_ptr+MACRO_NMAX, getHomingWithFoodNum<n>(), 0, binary_op);;
        }
        else{
            return getHomingFood<n>()(i);
        }
    }
};

template <> struct getHomingFood<0>{
    int operator()(int i){
        return -1;
    }
};

template <AntCharacter ch,int max> struct getHomingTypeAndFood{
    int operator()(int i){
        static const int n = max-1;
        static thrust::plus<int> binary_op;
        if(n==i){
            return thrust::transform_reduce(ants_d_ptr, ants_d_ptr+MACRO_NMAX, getHomingWithTypeAndFoodNum<ch,n>(), 0, binary_op);;
        }
        else{
            return getHomingTypeAndFood<ch,n>()(i);
        }
    }
};

template <AntCharacter ch> struct getHomingTypeAndFood<ch,0>{
    int operator()(int i){
        return -1;
    }
};

struct getPheroAroundFood{
    __device__ double operator()(const Food food) const{
        int i = food.i;
        int j = food.j;
        double sum = 0.0;
        sum += cells_d[i][j].phero;

        Cell *c=NULL;
        for(enum Direction dir = UP; dir<=UPLEFT; (dir<<=1) ){
            c = getCell(cells_d,i,j,dir);
            sum += c->phero;
        }
        return sum/7.0;
    }
};

// template <class T> std::string toString(T t){
//     static std::ostringstream ss;
//     ss << (T)t;

//     std::string str(ss.str());
//     ss.str("");
//     ss.clear(std::stringstream::goodbit);

//     return str;
// }

void IOInit(){	

    struct stat st;

    pw_old = -1;

    std::string nmax = toString(MACRO_NMAX);
    std::string max = toString(MACRO_MAX);

    std::string initDir(nmax+"ants_"+max+"x"+max+"cells");
    if(stat(initDir.c_str(), &st) != 0){
#ifdef _WIN32
        _mkdir(initDir.c_str());
#else
        mkdir(initDir.c_str(), 0775);
#endif
    }

    std::string fnum = toString(MACRO_NUM_FOODS);

    std::string fNumDir(initDir+"/"+fnum+"foodnum");
    if(stat(fNumDir.c_str(), &st) != 0){
#ifdef _WIN32
        _mkdir(fNumDir.c_str());
#else
        mkdir(fNumDir.c_str(), 0775);
#endif
    }

    std::string fsource = toString(MACRO_FOODSOURCE);
    std::string fdist = toString(MACRO_FOOD_DIST);

    std::string fCondDir(fNumDir+"/"+fsource+"initfvol_"+fdist+"fdist");
    if(stat(fCondDir.c_str(), &st) != 0){
#ifdef _WIN32
        _mkdir(fCondDir.c_str());
#else
        mkdir(fCondDir.c_str(), 0775);
#endif
    }

    std::string step = toString(MACRO_MAX_STEP);
    std::string angle = toString(MACRO_FOOD_ANGLE);

    std::string stepAngleDir(fCondDir+"/"+step+"steps_"+angle+"deg");
    if(stat(stepAngleDir.c_str(), &st) != 0){
#ifdef _WIN32
        _mkdir(stepAngleDir.c_str());
#else
        mkdir(stepAngleDir.c_str(), 0775);
#endif
    }

    path = std::string(stepAngleDir+"/");
    std::string rec = toString(MACRO_REC);
    fool_num_plot = std::string(path+step+"steps_"+angle+"deg_"+rec+"rec.dat");

    ofs = new std::ofstream(fool_num_plot.c_str());

    fool_num_plot_prob = std::string(path+step+"steps_"+angle+"deg_"+rec+"rec_prob.dat");
    ofsProb = new std::ofstream(fool_num_plot_prob.c_str());

    std::ifstream constifs(constHeader.c_str());
    std::ofstream constofs((path+constHeader).c_str());
    constofs << constifs.rdbuf() << std::flush;


    // fool_num_food = std::string(path+step+"steps_"+angle+"deg_food.dat");
    // ofsfood = new std::ofstream(fool_num_food.c_str());
}

void IOEffWrite(int pw, int n, double sum){
    if(pw_old<pw){
        pw_old = pw;
        (*ofs) << std::endl;
    }
    (*ofs)  << pw << " "
        << n  << " "
        << (sum/(MACRO_MAX_STEP))/(MACRO_MAX_TIME-1000)
        << std::endl;
}

void IOProbWrite(int pw, int n, double prob[]){
    if(pw_old<pw){
        pw_old = pw;
        (*ofsProb) << std::endl;
    }
    (*ofsProb)  << pw << " "
        << n  << " "
        << ((double)prob[0]/(MACRO_MAX_STEP))/(MACRO_MAX_TIME) << " "
        << ((double)prob[1]/(MACRO_MAX_STEP))/(MACRO_MAX_TIME) << " "
        << ((double)prob[2]/(MACRO_MAX_STEP))/(MACRO_MAX_TIME) << " "
        << ((double)prob[3]/(MACRO_MAX_STEP))/(MACRO_MAX_TIME) << " "
        << ((double)prob[4]/(MACRO_MAX_STEP))/(MACRO_MAX_TIME)
        << std::endl;
}

void IOFinWrite(int pw, int n, double sum){
    if(pw_old<pw){
        pw_old = pw;
        (*ofs) << std::endl;
    }
    (*ofs)  << pw << " "
        << n  << " "
        << (sum/(MACRO_MAX_STEP))
        << std::endl;
}

void IOFoodWrite(int pw, int n, double ft[], double et[]){
    if(pw_old<pw){
        pw_old = pw;
        (*ofsfood) << std::endl;
    }
    (*ofsfood)  << pw << " " << n  << " ";

    for(int id=0; id<MACRO_NUM_FOODS; id++)
        (*ofsfood) << ft[id]/(MACRO_MAX_STEP) << " ";

    for(int id=0; id<MACRO_NUM_FOODS; id++)
        for(int id2=id+1; id2<MACRO_NUM_FOODS; id2++)
            (*ofsfood) << (ft[id] - ft[id2])/(MACRO_MAX_STEP) << " ";

    for(int id=0; id<MACRO_NUM_FOODS; id++)
        (*ofsfood) << et[id]/(MACRO_MAX_STEP) << " ";

    for(int id=0; id<MACRO_NUM_FOODS; id++)
        for(int id2=id+1; id2<MACRO_NUM_FOODS; id2++)
            (*ofsfood) << (et[id] - et[id2])/(MACRO_MAX_STEP) << " ";

    (*ofsfood) << std::endl;
}

void IOFoodAmountWrite(std::ofstream & ofs, std::vector< std::vector<double> > & ft){
    // if(pw_old<pw){
    //     pw_old = pw;
    //     (*ofs) << std::endl;
    // }

    for(int t=0; t<MACRO_MAX_TIME; t++){
        (ofs)  << t << "\t" ;
        for(int id=0; id<MACRO_NUM_FOODS; id++)
            (ofs) << ft[t][id] << "\t";
        (ofs) << std::endl;
    }
}


void IOCellWrite(int pw, int n){
    std::string pwstr = toString(pw);
    std::string nstr = toString(n);
    std::string anglestr = toString(MACRO_FOOD_ANGLE);

    std::string celldata(path+"cell_"+anglestr+"deg_10e-"+pwstr+"_"+nstr+"normal"+".dat");
    std::ofstream cellfs(celldata.c_str());

    hipMemcpyFromSymbol(cells,HIP_SYMBOL(cells_d),MACRO_MAX*MACRO_MAX*sizeof(Cell),0);
    for(int i=0; i<MACRO_MAX; i++){
        for(int j=0; j<MACRO_MAX; j++){
            cellfs << cells[j][i].cart.x << " "
                << cells[j][i].cart.y << " "
                << cells[j][i].phero
                << std::endl;
        }
        cellfs << std::endl;
    }
}

void IOEffPoll(int pw, int n, int sample, int t){

    static getHomingWithType<NORMAL_CH> normalOp;
    static getHomingWithType<FOOL_CH> ahoOp;
    static thrust::plus<int> binary_op;
    static int homingFoods[MACRO_NUM_FOODS];
    static int foolHomingFoods[MACRO_NUM_FOODS];
    static int normalHomingFoods[MACRO_NUM_FOODS];
    static getHomingFood<MACRO_NUM_FOODS> homingFoodFunctor;
    static getHomingTypeAndFood<FOOL_CH,MACRO_NUM_FOODS> foolHomingFoodFunctor;
    static getHomingTypeAndFood<NORMAL_CH,MACRO_NUM_FOODS> normalHomingFoodFunctor;
    static thrust::host_vector<double> phero_h(MACRO_NUM_FOODS);
    static thrust::device_vector<double> phero_d(MACRO_NUM_FOODS);	

    std::string stepstr = toString(MACRO_MAX_STEP);
    std::string anglestr = toString(MACRO_FOOD_ANGLE);

    std::string pwstr = toString(pw);
    std::string nstr = toString(n);
    std::string samplestr = toString(sample);

    std::string celldata(path+"food_"+anglestr+"deg_10e-"+pwstr+"_"+nstr+"normal"+"_sampleNo"+samplestr+"_of_"+stepstr+".dat");
    std::ofstream pollfs(celldata.c_str(),std::ios::out | std::ios::app);

    int nor = thrust::transform_reduce(ants_d_ptr, ants_d_ptr+MACRO_NMAX, normalOp, 0, binary_op);
    int aho = thrust::transform_reduce(ants_d_ptr, ants_d_ptr+MACRO_NMAX, ahoOp, 0, binary_op);

    thrust::transform(foods_d_ptr, foods_d_ptr+MACRO_NUM_FOODS, phero_d.begin(), getPheroAroundFood());
    thrust::copy(phero_d.begin(), phero_d.end(), phero_h.begin());

    pollfs  << t << " ";

    for (int i=0; i<MACRO_NUM_FOODS; i++){
        //homingFoods[i] = homingFoodFunctor(i);
        foolHomingFoods[i] = foolHomingFoodFunctor(i);
        normalHomingFoods[i] = normalHomingFoodFunctor(i);
        homingFoods[i]=foolHomingFoods[i]+normalHomingFoods[i];
    }

    for (int i=0; i<MACRO_NUM_FOODS; i++){
        pollfs  << homingFoods[i]
            << " ";
    }
    pollfs  << nor      << " "
        << aho      << " "
        << nor/(double)n << " "
        << aho/(double)(MACRO_NMAX-n) << " "
        << (nor+aho)<< " ";
    for (int i=0; i<MACRO_NUM_FOODS; i++){
        pollfs << normalHomingFoods[i] << " ";
    }
    for (int i=0; i<MACRO_NUM_FOODS; i++){
        pollfs << foolHomingFoods[i] << " ";
    }
    for (int i=0; i<MACRO_NUM_FOODS; i++){
        pollfs << normalHomingFoods[i]/(double)n << " ";
    }
    for (int i=0; i<MACRO_NUM_FOODS; i++){
        pollfs << foolHomingFoods[i]/(double)(MACRO_NMAX-n) << " ";
    }
    pollfs << (nor+aho) << " ";
    for (int i=0; i<MACRO_NUM_FOODS; i++){
        pollfs << phero_h[i] << " ";
    }
    pollfs << std::endl;
}
