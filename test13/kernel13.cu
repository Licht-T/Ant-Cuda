#include "hip/hip_runtime.h"
#define _USE_MATH_DEFINES

#include "kernel13.h"

__constant__ double SENSOR2;
__constant__ int NAHO;

__device__ int selectedCounts[MACRO_NMAX];
__device__ double tmpPhero_d[MACRO_MAX][MACRO_MAX];
__device__ hiprandState rnd_state[MACRO_NMAX];

//Misc
__device__ bool isGotFood(Food& food);
__device__ double atomicAddDouble(double* address, double val);
__device__ enum Direction genDirRand(int id);
__device__ double genProbRand(int id);
__device__ int genAntNumRand(int id);
__device__ double degToRad(double a);
__device__ double dist(Cell a,Cell b);
__device__ double distCandP(Cell a,double x,double y);
__device__ bool isOppositeDir(enum Direction nestDir,enum Direction dir);
__device__ bool isOppositeDir(Cell& cell, enum Direction dir);
__device__ enum Direction selectNextDir(Cell& cell, enum Direction dir);
__device__ double hilFunc(double x,double alpha);

//Initializer
__host__ void getDevicePtrs();
__global__ void randInit();
__global__ void antsInit();
__global__ void cellsInit();
__global__ void setNest();
__global__ void setDistFromNest();
__global__ void setNestDirs();
__global__ void setNearestDirFromNest();
__global__ void setFoodsDir();

//Calculation functions
__global__ void selectAnts();
__global__ void naturalFoodDecrease();
__global__ void evapolation();
__global__ void chemotaxis();
__global__ void diffusion();
__global__ void pheroUpdate();


__host__ void calculation(){
    naturalFoodDecrease<<<1,MACRO_NUM_FOODS>>>();
    evapolation<<<MACRO_MAX,MACRO_MAX>>>();

    //sortKeyInit<<<1,MACRO_NMAX>>>();
    //thrust::sort_by_key(sort_key_d_ptr, sort_key_d_ptr + MACRO_NMAX, ants_d_ptr);

    selectAnts<<<1,MACRO_NMAX>>>();
    chemotaxis<<<1,MACRO_NMAX>>>();
    //hipMemcpyFromSymbol(cells,HIP_SYMBOL(cells_d),MACRO_MAX*MACRO_MAX*sizeof(Cell),0);
    //chemotaxis();
    //hipMemcpyToSymbol(HIP_SYMBOL(cells_d),cells,MACRO_MAX*MACRO_MAX*sizeof(Cell),0);
    diffusion<<<MACRO_MAX,MACRO_MAX>>>();
    pheroUpdate<<<MACRO_MAX,MACRO_MAX>>>();
}

//Initialize

__host__ void getDevicePtrs(){
    hipGetSymbolAddress((void**)&sort_key_d_ptr_raw, sort_key_d);
    sort_key_d_ptr = thrust::device_ptr<unsigned int>(sort_key_d_ptr_raw);

    hipGetSymbolAddress((void**)&seeds_d_ptr_raw, seeds_d);
    seeds_d_ptr = thrust::device_ptr<unsigned long long int>(seeds_d_ptr_raw);

    hipGetSymbolAddress((void**)&ants_d_ptr_raw, ants_d);
    ants_d_ptr = thrust::device_ptr<Ant>(ants_d_ptr_raw);

    hipGetSymbolAddress((void**)&cells_d_ptr_raw, cells_d);
    cells_d_ptr = thrust::device_ptr<Cell>(cells_d_ptr_raw);

    hipGetSymbolAddress((void**)&foods_d_ptr_raw, foods_d);
    foods_d_ptr = thrust::device_ptr<Food>(foods_d_ptr_raw);
}

__global__ void randInit(){
    const int id = threadIdx.x + blockIdx.x * blockDim.x;
    hiprand_init(seeds_d[id],0,0,&rnd_state[id]);
}

__global__ void antsReset(){
    const int id = threadIdx.x + blockIdx.x * blockDim.x;
    ants_d[id].status = FORAGE;
    ants_d[id].i = MACRO_NEST_Y;
    ants_d[id].j = MACRO_NEST_X;
    ants_d[id].searchTime = 0;
    ants_d[id].dir = genDirRand(id);
    for (int i=0; i<MACRO_NUM_FOODS; i++){
        ants_d[id].homing[i] = 0;
    }
    if(id<NAHO){
        ants_d[id].ch = FOOL_CH;
    }
    else {
        ants_d[id].ch = NORMAL_CH;
    }
}

__global__ void cellsReset(){
    const int i = threadIdx.x;
    const int j = blockIdx.x;
    cells_d[i][j].phero = 0.0;
}

__global__ void cellsInit(){
    const int i = threadIdx.x;
    const int j = blockIdx.x;
    cells_d[i][j].foodNo = -1;
    cells_d[i][j].status = NORMAL_CELL;

    //Cell number initialize
    cells_d[i][j].i = i;
    cells_d[i][j].j = j;

    //Cartesian initialize
    cells_d[i][j].cart.x = (j-MACRO_CART_X_ZERO)*(sqrt(3.0)/2.0);
    cells_d[i][j].cart.y = (abs(j-MACRO_CART_X_ZERO)%2)/2.0+(i-MACRO_CART_Y_ZERO);
    //Edge initialize
    cells_d[i][j].edge = NONE;

    //Nest Dir initialize
    cells_d[i][j].nestDir = NONE;

    cells_d[i][j].distFromNest = 0.0;
}


__global__ void setEdges(){
    const int i = threadIdx.x;
    const int j = blockIdx.x;
    if(i==MACRO_MAX-1){ //For upper edge
        cells_d[i][j].edge |= UP;
        if(abs((j-MACRO_CART_X_ZERO)%2)==1){
            cells_d[i][j].edge |= (UPLEFT | UPRIGHT);
        }
    }
    else if(i==0){//For lower edge
        cells_d[i][j].edge |= LOW;
        if(abs((j-MACRO_CART_X_ZERO)%2)==0){
            cells_d[i][j].edge |= LOWLEFT | LOWRIGHT;
        }
    }

    if(j==0){//For left edge
        cells_d[i][j].edge |= LEFT;
    }
    else if(j==MACRO_MAX-1){//For right edge
        cells_d[i][j].edge |= RIGHT;
    }
}

__global__ void setNest(){
    const int i = threadIdx.x;
    const int j = blockIdx.x;

    Cell* c;
    if(i==MACRO_NEST_Y && j==MACRO_NEST_X){
        cells_d[MACRO_NEST_Y][MACRO_NEST_X].status |= NEST_CELL;

        for(enum Direction d = UP; d<=UPLEFT; (d<<=1) ){
            c = getCell(cells_d,MACRO_NEST_Y,MACRO_NEST_X,d);
            c->status |= NEST_NEIGHBOUR_CELL;
        }
    }
}

__global__ void setDistFromNest(){
    const int i = threadIdx.x;
    const int j = blockIdx.x;

    Cell *nest_c;
    nest_c = &cells_d[MACRO_NEST_Y][MACRO_NEST_X];
    double d = dist(cells_d[i][j],*nest_c);
    cells_d[i][j].distFromNest = d;
}

__device__ double dot(Cartesian a, Cartesian b) {
    return (a.x * b.x + a.y * b.y);
}

__device__ double cross(Cartesian a, Cartesian b) {
    return (a.x * b.y - a.y * b.x);
}

__global__ void setCriticalAngle() {
    const int i = threadIdx.x;
    const int j = blockIdx.x;


    cells_d[i][j].criticalAngle = NONE;

    if( (cells_d[i][j].status&NEAR_NEST)!=NORMAL_CELL ){
        return;
    }

    Cartesian c = cells_d[i][j].cart;
    c.x = -c.x/cells_d[i][j].distFromNest;
    c.y = -c.y/cells_d[i][j].distFromNest;

    for(enum Direction dir = UP; dir<=UPLEFT; (dir<<=1) ) {
        Cartesian d;

        switch (dir) {
            case UP:
                d.x = 0;
                d.y = 1;
                break;
            case UPRIGHT:
                d.x = 1;
                d.y = tan(M_PI/4.0);
                break;
            case LOWRIGHT:
                d.x = 1;
                d.y = -tan(M_PI/4.0);
                break;
            case LOW:
                d.x = 0;
                d.y = -1;
                break;
            case LOWLEFT:
                d.x = -1;
                d.y = -tan(M_PI/4.0);
                break;
            case UPLEFT:
                d.x = -1;
                d.y = tan(M_PI/4.0);
                break;
            default:
                break;
        }

        d.x = d.x/sqrt(dot(d,d));
        d.y = d.y/sqrt(dot(d,d));

        double dotVal = dot(c,d);
        if (dotVal<=0.3){
            cells_d[i][j].criticalAngle |= dir;
        }
    }
}


__global__ void setNearestDirFromNest(){
    const int i = threadIdx.x;
    const int j = blockIdx.x;
    Cell& c = cells_d[i][j];

    for (int itr=0; itr<6; itr++){
        c.nearestDirFromNestList[itr] = NONE;
    }

    enum Direction dir = UP;
    for(int itr=0; dir<=UPLEFT; itr++) {
        if ( c.criticalAngle&dir == NONE ){
            continue;
        }

        c.nearestDirFromNestList[itr] = selectNextDir(c, dir);
        dir<<=1;
    }
}

__global__ void setNestDirs(){
    const int i = threadIdx.x;
    const int j = blockIdx.x;

    Cell *c;

    double d = cells_d[i][j].distFromNest;
    double tmp;
    for(enum Direction dir = UP; dir<=UPLEFT; (dir<<=1) ){

        c = getCell(cells_d,i,j,dir);


        tmp=c->distFromNest;
        if( fabs(tmp-d)<MACRO_EPS ){
            cells_d[i][j].nestDir |= dir;
        }
        else if(tmp<d) {
            cells_d[i][j].nestDir = dir;
            d = tmp;
        }
    }
}

__global__ void foodsReset(){
    const int i = threadIdx.x + blockIdx.x * blockDim.x;
    foods_d[i].vol = MACRO_FOODSOURCE;
}

__global__ void setFoodsDir(){
    const int i = threadIdx.x + blockIdx.x * blockDim.x;
    const double dtheta = degToRad(MACRO_FOOD_ANGLE);

    Cell *nearCell=NULL;

    double x,y;
    x=MACRO_FOOD_DIST * cos(i*dtheta);
    y=MACRO_FOOD_DIST * sin(i*dtheta);
    for(int j=0; j<MACRO_MAX; j++){
        for(int k=0; k<MACRO_MAX; k++){
            if(distCandP(cells_d[j][k],x,y)<=sqrt(3.0)/3.0+MACRO_EPS){
                nearCell = &cells_d[j][k];
                break;
            }
        }
    }
    if(nearCell==NULL){
    }
    else{
        Cell *c=NULL;
        double d = distCandP(*nearCell,x,y);
        int j = nearCell->i;
        int k = nearCell->j;

        for(enum Direction dir = UP; dir<=UPLEFT; (dir<<=1) ){
            c = getCell(cells_d,j,k,dir);
            if( distCandP(*c,x,y)<d ){
                nearCell = c;
                d = distCandP(*nearCell,x,y);
            }
        }
        foods_d[i].i = nearCell->i;
        foods_d[i].j = nearCell->j;

        nearCell->foodNo = i;
        nearCell->status |= FOOD_CELL;


        for(enum Direction dir = UP; dir<=UPLEFT; (dir<<=1) ){
            c = getCell(cells_d,foods_d[i].i,foods_d[i].j,dir);
            c->foodNo = i;
            c->status |= FOOD_NEIGHBOUR_CELL;
        }
    }

}


//Calculation

__global__ void selectAnts(){
    const int id = threadIdx.x + blockIdx.x * blockDim.x;
    int rnd =  genAntNumRand(id);
    atomicAdd(&selectedCounts[rnd], 1);
}

__global__ void sortKeyInit(){
    const int id = threadIdx.x + blockIdx.x * blockDim.x;
    sort_key_d[id] = hiprand(&rnd_state[id]);
    //printf("id:%d,%u\n",id,sort_key_d[id]);
}

__global__ void diffusion(){
    const int i = blockIdx.x;
    const int j = threadIdx.x;

    double tmp = 0.0;
    for (enum Direction dir = UP; dir<=UPLEFT; (dir<<=1) ){
        tmp += getCell(cells_d,i,j,dir)->phero;
    }
    tmpPhero_d[i][j] = cells_d[i][j].phero+MACRO_DIFFE*(tmp/6.0-cells_d[i][j].phero);
}

__global__ void pheroUpdate(){
    const int i = blockIdx.x;
    const int j = threadIdx.x;

    cells_d[i][j].phero = tmpPhero_d[i][j];
}

__global__ void naturalFoodDecrease(){
    const int id = threadIdx.x + blockIdx.x * blockDim.x;
    // 常に一定量で増加する。
    foods_d[id].vol = ( foods_d[id].vol + 1.0 ) * MACRO_REC;
    if (foods_d[id].vol > 100.0)
        foods_d[id].vol = 100.0;
}

__global__ void evapolation(){
    const int i = blockIdx.x;
    const int j = threadIdx.x;
    cells_d[i][j].phero *= (1.0-MACRO_EVAPOLATION_CONST);
}


__global__ void chemotaxis(){
    const int id = threadIdx.x + blockIdx.x * blockDim.x;
    Ant *ant = &(ants_d[id]);

    for(int dummy=0; dummy<selectedCounts[id]; dummy++){
        ant->searchTime++;

        int i = ant->i;
        int j = ant->j;
        enum Direction dir = ant->dir;
        enum Direction nestDir = cells_d[i][j].nestDir;

        double leftPhero, frontPhero, rightPhero;

        Cell *leftCell  = getCell(cells_d,i,j,left(dir));
        Cell *frontCell = getCell(cells_d,i,j,dir);
        Cell *rightCell = getCell(cells_d,i,j,right(dir));

        if(
                ant->searchTime>=MACRO_MAX_SEARCH_TIME
                && ant->status!=EMERGENCY
          ){
            ant->status = EMERGENCY;
        }

        if(ant->status==GOHOME){
            atomicAddDouble(&(cells_d[i][j].phero),MACRO_EMI*MACRO_ENEST);
        }
        __threadfence();
        if(ant->status==RANDOM_SEARCH){
            leftPhero = 1.0;
            frontPhero = 1.0;
            rightPhero = 1.0;
        }
        else {
            leftPhero = leftCell->phero;
            frontPhero = frontCell->phero;
            rightPhero = rightCell->phero;
        }

        if( (ant->status==GOHOME || ant->status==EMERGENCY) && isOppositeDir(cells_d[i][j], dir)){

            enum Direction nextDir = cells_d[i][j].nearestDirFromNestList[dirToNum(dir)];

            if( nextDir == left(dir) ){
                ant->dir = left(dir);
                frontCell = leftCell;
            }
            else if( nextDir == right(dir) ){
                ant->dir = right(dir);
                frontCell = rightCell;
            }
            else{
                if(genProbRand(id)<=0.5){
                    ant->dir = right(dir);
                    frontCell = rightCell;
                }
                else{
                    ant->dir = left(dir);
                    frontCell = leftCell;
                }
            }
            ant->i = frontCell->i;
            ant->j = frontCell->j;
        }
        else{
            double s1,s2,s3,s12,t,tot,rand;
            if(ant->ch == NORMAL_CH){
                t = MACRO_HIL_CONST;
            }
            else{
                t = SENSOR2*MACRO_HIL_CONST;
            }

            s1=hilFunc(leftPhero,t);
            s2=hilFunc(frontPhero,t);
            s3=hilFunc(rightPhero,t);
            /*
               if(s1<MACRO_EPS && s2<MACRO_EPS && s3<MACRO_EPS){
               s1=1.0;
               s2=1.0;
               s3=1.0;
               }
               */
            tot = s1+s2+s3;
            s1/=tot;
            s2/=tot;

            s12=s1+s2;

            rand=genProbRand(id);

            if(rand<=s1){
                ant->dir = left(dir);
                ant->i   = leftCell->i;
                ant->j   = leftCell->j;
            }
            else if(rand<=s12){
                ant->i   = frontCell->i;
                ant->j   = frontCell->j;
            }
            else{
                ant->dir = right(dir);
                ant->i   = rightCell->i;
                ant->j   = rightCell->j;
            }

        }

        if( (cells_d[ant->i][ant->j].status&NEAR_FOOD)!=NORMAL_CELL
                &&  foods_d[  cells_d[ant->i][ant->j].foodNo  ].vol>=0.1
                &&  (ant->status != GOHOME && ant->status != EMERGENCY) ){
            //atomicAddDouble(&(foods_d[  cells_d[ant->i][ant->j].foodNo  ].vol),-MACRO_UNIT);
            //ant->status = GOHOME;
            //ant->searchTime = 0;
            int fNo = cells_d[ant->i][ant->j].foodNo;

            if(isGotFood(foods_d[fNo])){
                ant->status = GOHOME;
                ant->searchTime = 0;
                ant->_foodNo = fNo;
                ant->dir = left(left(left(dir)));
            }
        }
        __threadfence();

        if( (cells_d[ant->i][ant->j].status&NEAR_NEST)!=NORMAL_CELL
                &&  (ant->status == GOHOME || ant->status == EMERGENCY)){
            if(ant->status == GOHOME){
                ant->homing[ant->_foodNo]++;
                //atomicAddDouble(&(cells_d[i][j].phero),MACRO_EMI*MACRO_ENEST);
            }
            ant->status = FORAGE;
            ant->searchTime = 0;
            ant->dir = genDirRand(id);
            ant->i   = MACRO_NEST_Y;
            ant->j   = MACRO_NEST_X;
        }
    }
    selectedCounts[id] = 0;
}


//DataHandler
__device__ __host__  enum Direction operator<<(enum Direction d, int i){
    return static_cast<enum Direction>(static_cast<int>(d)<<i);
}

__device__ __host__  enum Direction operator>>(enum Direction d, int i){
    return static_cast<enum Direction>(static_cast<int>(d)>>i);
}

__device__ __host__  enum Direction operator|(enum Direction d1, enum Direction d2){
    return static_cast<enum Direction>(static_cast<int>(d1)|static_cast<int>(d2));
}
__device__ __host__  enum Direction operator&(enum Direction d1, enum Direction d2){
    return static_cast<enum Direction>(static_cast<int>(d1)&static_cast<int>(d2));
}

__device__ __host__  enum Direction& operator|=(enum Direction& d1, enum Direction d2){
    d1 = (d1 | d2);
    return d1;
}

__device__ __host__  enum Direction& operator&=(enum Direction& d1, enum Direction d2){
    d1 = (d1 & d2);
    return d1;
}

__device__ __host__  enum Direction& operator<<=(enum Direction& d1, int i){
    d1 = (d1 << i);
    return d1;
}

__device__ __host__  enum Direction& operator>>=(enum Direction& d1, int i){
    d1 = (d1 >> i);
    return d1;
}

__device__ __host__  bool operator<=(enum Direction d1, enum Direction d2){
    return (static_cast<int>(d1) <= static_cast<int>(d2));
}







__device__ __host__  enum CELLStatus operator<<(enum CELLStatus d, int i){
    return static_cast<enum CELLStatus>(static_cast<int>(d)<<i);
}

__device__ __host__  enum CELLStatus operator>>(enum CELLStatus d, int i){
    return static_cast<enum CELLStatus>(static_cast<int>(d)>>i);
}

__device__ __host__  enum CELLStatus operator|(enum CELLStatus d1, enum CELLStatus d2){
    return static_cast<enum CELLStatus>(static_cast<int>(d1)|static_cast<int>(d2));
}
__device__ __host__  enum CELLStatus operator&(enum CELLStatus d1, enum CELLStatus d2){
    return static_cast<enum CELLStatus>(static_cast<int>(d1)&static_cast<int>(d2));
}

__device__ __host__  enum CELLStatus& operator|=(enum CELLStatus& d1, enum CELLStatus d2){
    d1 = (d1 | d2);
    return d1;
}

__device__ __host__  enum CELLStatus& operator&=(enum CELLStatus& d1, enum CELLStatus d2){
    d1 = (d1 & d2);
    return d1;
}



__device__ __host__ __forceinline__ enum Direction left(enum Direction dir){
    if(dir == UP){
        return UPLEFT;
    }
    else{
        return (dir >> 1)&ALL_DIR;
    }
}

__device__ __host__ __forceinline__ enum Direction right(enum Direction dir){
    if(dir == UPLEFT){
        return UP;
    }
    else{
        return (dir << 1)&ALL_DIR;
    }
}

__device__ __host__ __forceinline__ Cell* up(Cell cells[MACRO_MAX][MACRO_MAX],int i,int j){
    if( (cells[i][j].edge&UP)!=NONE ){
        return &cells[0][j];
    }
    else{
        return &cells[i+1][j];
    }
}

__device__ __host__ __forceinline__ Cell* upright(Cell cells[MACRO_MAX][MACRO_MAX],int i,int j){
    int ii,jj;
    if( (cells[i][j].edge&UPRIGHT)!=NONE ){
        jj = 0;
        if(abs(j-MACRO_CART_X_ZERO)%2==0){
            ii = i;
        }
        else{
            ii = i+1;
            if(ii==MACRO_MAX){
                ii = 0;
            }
        }
    }
    else{
        jj = j+1;
        if(abs(j-MACRO_CART_X_ZERO)%2==0){
            ii = i;
        }
        else{
            ii = i+1;
        }
    }
    return &cells[ii][jj];
}

__device__ __host__ __forceinline__ Cell* lowright(Cell cells[MACRO_MAX][MACRO_MAX],int i,int j){

    int ii,jj;

    if( (cells[i][j].edge&LOWRIGHT)!=NONE ){
        jj = 0;
        if(abs(j-MACRO_CART_X_ZERO)%2==0){
            ii = i-1;
            if(ii<0){
                ii=MACRO_MAX-1;
            }
        }
        else{
            ii = i;
        }
    }
    else{
        jj = j+1;
        if(abs(j-MACRO_CART_X_ZERO)%2==0){
            ii = i-1;
        }
        else{
            ii = i;
        }
    }
    return &cells[ii][jj];
}

__device__ __host__ __forceinline__ Cell* low(Cell cells[MACRO_MAX][MACRO_MAX],int i,int j){
    if( (cells[i][j].edge&LOW)!=NONE ){
        return &cells[MACRO_MAX-1][j];
    }
    else{
        return &cells[i-1][j];
    }
}

__device__ __host__ __forceinline__ Cell* lowleft(Cell cells[MACRO_MAX][MACRO_MAX],int i,int j){
    int ii,jj;

    if( (cells[i][j].edge&LOWLEFT)!=NONE ){
        jj = MACRO_MAX-1;
        if(abs(j-MACRO_CART_X_ZERO)%2==0){
            ii = i-1;
            if(ii<0){
                ii = MACRO_MAX-1;
            }
        }
        else{
            ii = i;
        }
    }
    else{
        jj = j-1;
        if(abs(j-MACRO_CART_X_ZERO)%2==0){
            ii = i-1;
        }
        else{
            ii=i;
        }
    }
    return &cells[ii][jj];
}

__device__ __host__ __forceinline__ Cell* upleft(Cell cells[MACRO_MAX][MACRO_MAX],int i,int j){
    int ii,jj;
    if( (cells[i][j].edge&UPLEFT)!=NONE ){
        jj = MACRO_MAX-1;

        if(abs(j-MACRO_CART_X_ZERO)%2==0){
            ii = i;
        }
        else{
            ii= i+1;
            if(ii==MACRO_MAX){
                ii=0;
            }
        }
    }
    else{
        jj = j-1;
        if(abs(j-MACRO_CART_X_ZERO)%2==0){
            ii = i;
        }
        else{
            ii = i+1;
        }
    }
    return &cells[ii][jj];
}

__device__ __host__ Cell* getCell(Cell cells[MACRO_MAX][MACRO_MAX],int i,int j, enum Direction dir){

    switch (dir){
        case UP:
            return up(cells,i,j);
        case UPRIGHT:
            return upright(cells,i,j);
        case LOWRIGHT:
            return lowright(cells,i,j);
        case LOW:
            return low(cells,i,j);
        case LOWLEFT:
            return lowleft(cells,i,j);
        case UPLEFT:
            return upleft(cells,i,j);
        default:
            return NULL;
    }
}

__device__ __host__ int dirToNum(enum Direction dir){
    switch (dir){
        case UP:
            return 0;
        case UPRIGHT:
            return 1;
        case LOWRIGHT:
            return 2;
        case LOW:
            return 3;
        case LOWLEFT:
            return 4;
        case UPLEFT:
            return 5;
        default:
            return -1;
    }
}



//Misc
__device__ __forceinline__ bool isGotFood(Food& food){
    unsigned long long int* address_as_ull =
        (unsigned long long int*)(&(food.vol));
    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        if(__longlong_as_double(assumed)<0.1){
            return false;
        }
        old = atomicCAS(address_as_ull, assumed,__double_as_longlong(-MACRO_UNIT + __longlong_as_double(assumed)));
    } while (assumed != old);
    return true;
}

__device__ __forceinline__ double atomicAddDouble(double* address, double val){
    unsigned long long int* address_as_ull =
        (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                __double_as_longlong(val +
                    __longlong_as_double(assumed)));
    } while (assumed != old);
    return __longlong_as_double(old);
}

__device__ __forceinline__ enum Direction genDirRand(int id){
    return static_cast<enum Direction>(1 << (hiprand(&rnd_state[id])%6));
}

__device__ __forceinline__ double genProbRand(int id){
    return hiprand_uniform_double(&rnd_state[id]);
}

__device__ __forceinline__ int genAntNumRand(int id){
    return hiprand(&rnd_state[id])%MACRO_NMAX;
}

__device__ __forceinline__ double degToRad(double a) {
    return a * M_PI / 180.0;
}

__device__ __forceinline__ double dist(Cell a,Cell b){
    return sqrt( (a.cart.x - b.cart.x)*(a.cart.x - b.cart.x)
            + (a.cart.y - b.cart.y)*(a.cart.y - b.cart.y) );
}

__device__ __forceinline__ double distCandP(Cell a,double x,double y){
    return sqrt( (a.cart.x - x)*(a.cart.x - x)
            + (a.cart.y - y)*(a.cart.y - y) );
}

__device__ __forceinline__ bool isOppositeDir(enum Direction nestDir,enum Direction dir){
    //If theta = 60 deg., this is OK.
    if( (dir&nestDir)        !=NONE
            ||  (left(dir)&nestDir)  !=NONE
            ||  (right(dir)&nestDir) !=NONE){
        return false;
    }
    else{
        return true;
    }
}

__device__ __forceinline__ bool isOppositeDir(Cell& cell, enum Direction dir){
    if ( (cell.criticalAngle & dir)==dir ){
        return true;
    }
    else{
        return false;
    }
}

__device__ __forceinline__ enum Direction selectNextDir(Cell& cell, enum Direction dir){
    int rightCount = 0;
    int leftCount  = 0;
    for (enum Direction currentDir=right(dir); currentDir!=dir; currentDir=right(currentDir)){
        if( (cell.criticalAngle & currentDir)!=currentDir ){
            break;
        }
        rightCount++;
    }

    for (enum Direction currentDir=left(dir); currentDir!=dir; currentDir=left(currentDir)){
        if( (cell.criticalAngle & currentDir)!=currentDir ){
            break;
        }
        leftCount++;
    }

    if ( rightCount < leftCount ){
        return right(dir);
    }
    else if ( rightCount > leftCount ){
        return left(dir);
    }
    else{
        return NONE;
    }
}

__device__ __forceinline__ double hilFunc(double x,double alpha){
    return pow(alpha*x+0.05,10);
}

__host__ void initialize(){
    getDevicePtrs();

    //antsInit<<<MACRO_NMAX,1>>>();
    cellsInit<<<MACRO_MAX,MACRO_MAX>>>();

    setEdges<<<MACRO_MAX,MACRO_MAX>>>();
    setNest<<<MACRO_MAX,MACRO_MAX>>>();
    setDistFromNest<<<MACRO_MAX,MACRO_MAX>>>();

    setCriticalAngle<<<MACRO_MAX,MACRO_MAX>>>();
    setNearestDirFromNest<<<MACRO_MAX,MACRO_MAX>>>();

    setNestDirs<<<MACRO_MAX,MACRO_MAX>>>();
    setFoodsDir<<<MACRO_NUM_FOODS,1>>>();
}

__host__ void reset(double sensor,int naho,unsigned long long int step){
    hipMemcpyToSymbol(HIP_SYMBOL(SENSOR2),&sensor,sizeof(double),0);
    hipMemcpyToSymbol(HIP_SYMBOL(NAHO),&naho,sizeof(int),0);

    //initialize();
    //antsInit<<<MACRO_NMAX,1>>>();
    //cellsInit<<<MACRO_MAX,MACRO_MAX>>>();

    //setEdges<<<MACRO_MAX,MACRO_MAX>>>();
    //setNest<<<MACRO_MAX,MACRO_MAX>>>();
    //setDistFromNest<<<MACRO_MAX,MACRO_MAX>>>();

    //setNestDirs<<<MACRO_MAX,MACRO_MAX>>>();
    //setFoodsDir<<<MACRO_NUM_FOODS,1>>>();

    srand(MACRO_RND_SEED+step);

    thrust::host_vector<unsigned long long int> seeds_vec_h(MACRO_NMAX);
    std::generate(seeds_vec_h.begin(), seeds_vec_h.end(), rand);
    thrust::copy(seeds_vec_h.begin(), seeds_vec_h.end(), seeds_d_ptr);
    randInit<<<MACRO_NMAX,1>>>();

    antsReset<<<MACRO_NMAX,1>>>();
    cellsReset<<<MACRO_MAX,MACRO_MAX>>>();
    foodsReset<<<MACRO_NUM_FOODS,1>>>();
}
