#include "hip/hip_runtime.h"
#include <iostream>
#include <sstream>
#include <fstream>
#include <cmath>

#include <sys/stat.h>
#include <sys/types.h>

// includes CUDA Runtime
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <thrust/transform_reduce.h>
#include <thrust/functional.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>

#include "Constants.h"
#include "DataStructures.h"
#include "Variables.h"
#include "kernel6.h"
#include "Display.h"
#include "IO6.h"

int main(int argc, char *argv[]){

    getHoming homeOp;
    thrust::plus<int> binary_op;

    IOInit();

    initialize();

    double normalSum = 0.0;

    int access[MACRO_NUM_FOODS] = {0, 0};
    double prob[5] = {0.0, 0.0, 0.0, 0.0, 0.0}; // 0:どちらにもアクセスしていない, 1:片方にアクセス, 2:両方にアクセス, 3:id=0にアクセスしている状態, 4:id=1にアクセスしている状態
    double probNormal[5] = {0.0, 0.0, 0.0, 0.0, 0.0};
    double foodspre[MACRO_NUM_FOODS] = {0.0, 0.0};

    /* for(unsigned long long int dummy=1; dummy<=MACRO_MAX_STEP; dummy++){ */
    /*     reset(pow(10,-3),50,dummy); */
    /*     display(argc,argv); */
    /*  */
    /*     for(int t=0; t<MACRO_MAX_TIME; t++){ */
    /*         calculation(); */
    /*     } */
    /* } */

    // 馬鹿ありのいない場合
    for(unsigned long long int dummy=1; dummy<=MACRO_MAX_STEP; dummy++){
        reset(1,0,dummy);

        // display(argc,argv);

        for (int id=0; id<MACRO_NUM_FOODS; id++)
            foodspre[id] = MACRO_FOODSOURCE;
        for(int t=0; t<MACRO_MAX_TIME; t++){
            calculation();
            hipMemcpyFromSymbol(&foods, HIP_SYMBOL(foods_d), sizeof(Food)*MACRO_NUM_FOODS);
            for (int id=0; id<MACRO_NUM_FOODS; id++){
                if( foods[id].vol < foodspre[id] + MACRO_REC ){
                    access[id] = 1;
                    probNormal[3+id] += 1;
                }
                else{
                    access[id] = 0;
                }
                foodspre[id] = foods[id].vol;
            }
            probNormal[access[0]+access[1]] += 1;
            // if(t%500==0){
            //     IOEffPoll(0,500,dummy,t);
            // }

        }

        normalSum += thrust::transform_reduce(ants_d_ptr, ants_d_ptr+MACRO_NMAX, homeOp, 0, binary_op);
    }

    for (int n=0; n<=500; n+=50){
        IOEffWrite(0,n,normalSum);
        IOProbWrite(0,n,probNormal);
    }

    // 馬鹿アリの感受性
    for (int pw=1; pw<=7; pw++){
        // 正常アリの数
        for (int n=0; n<=450; n+=50){
            double sensor = pow(10,-pw);
            int naho = (MACRO_NMAX - n);

            double sum = 0.0;
            for (int p=0; p<5; p++)
                prob[p] = 0;

            for(unsigned long long int dummy=1; dummy<=MACRO_MAX_STEP; dummy++){
                reset(sensor,naho,dummy);

                for (int id=0; id<MACRO_NUM_FOODS; id++)
                    foodspre[id] = MACRO_FOODSOURCE;

                for(int t=1; t<=MACRO_MAX_TIME; t++){
                    calculation();
                    hipMemcpyFromSymbol(&foods, HIP_SYMBOL(foods_d), sizeof(Food)*MACRO_NUM_FOODS);

                    for (int id=0; id<MACRO_NUM_FOODS; id++){
                        if( foods[id].vol < foodspre[id] + MACRO_REC ){
                            access[id] = 1;
                            prob[3+id] += 1;
                        }
                        else{
                            access[id] = 0;
                        }
                        foodspre[id] = foods[id].vol;
                    }
                    prob[access[0]+access[1]] += 1;
                    // if(t%500==0){
                    //     IOEffPoll(pw,n,dummy,t);
                    // }

                }

                sum += thrust::transform_reduce(ants_d_ptr, ants_d_ptr+MACRO_NMAX, homeOp, 0, binary_op);
            }
            // IOCellWrite(pw,n);
           IOEffWrite(pw,n,sum);
           IOProbWrite(pw,n,prob);
        }
        IOEffWrite(pw,500,normalSum);
        IOProbWrite(pw,500,probNormal);
    }

}

